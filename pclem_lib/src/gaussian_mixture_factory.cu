#include "hip/hip_runtime.h"

#include <cstdlib>
#include <ctime>
#include <glog/logging.h>

#include "point.h"
#include "gaussian_mixture_factory.h"

namespace pclem {
    bool GaussianMixtureFactory::random_seeded = false;

    GaussianMixtureFactory::GaussianMixtureFactory() {
        if(!random_seeded) {
            srand(static_cast <unsigned> (time(0)));
        }
    }

    GaussianMixture GaussianMixtureFactory::from_pcl_corners(const PointCloud& pcl) const {
        return from_pcl_corners(pcl.getBoundingBox());
    }

    GaussianMixture GaussianMixtureFactory::from_pcl_corners(const DevicePointCloud& pcl) const {
        return from_pcl_corners(pcl.getBoundingBox());
    }

    GaussianMixture GaussianMixtureFactory::from_pcl_corners(const BoundingBox& bounding_box) const {
        auto corners = bounding_box.corners();
        double initial_weight_of_gaussian = 1.0 / corners.size();

        std::vector<WeightedGaussian> temp_gaussians;
        for(Point corner : corners) {
            CovarianceMatrix sigma = CovarianceMatrix();
            sigma.set(0,0,10.0);
            sigma.set(1,1,10.0);
            sigma.set(2,2,10.0);

            WeightedGaussian gaussian(corner, sigma, initial_weight_of_gaussian);
            VLOG(4) << "Adding gaussian: " << gaussian;
            temp_gaussians.push_back(gaussian);
        }

        GaussianMixture mixture(temp_gaussians);
        return mixture;
    }

    GaussianMixture GaussianMixtureFactory::around_point(const Point& point, const CovarianceMatrix& cov, int n_of_distributions, double delta) const {
        VLOG(10) << "Creating gaussian mixture around point...";

        std::vector<WeightedGaussian> temp_gaussians;

        for(int i = 0; i < n_of_distributions; i++) {
            WeightedGaussian gaussian(
                Point(point.x + random_number(-delta, delta),
                      point.y + random_number(-delta, delta),
                      point.z + random_number(-delta, delta)),
                cov,
                1.0 / n_of_distributions);

            temp_gaussians.push_back(gaussian);
            std::cout << temp_gaussians.back();
        }

        GaussianMixture resulting_mixture(temp_gaussians);

        VLOG(10) << "Done creating gaussian mixture.";
        return resulting_mixture;
    }

    double GaussianMixtureFactory::random_number(double min, double max) {
        return min + static_cast<double> (rand()) / static_cast<double> (RAND_MAX/(max-min));
    }
}
