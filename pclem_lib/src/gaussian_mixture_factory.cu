#include "hip/hip_runtime.h"

#include <cstdlib>
#include <ctime>
#include <glog/logging.h>

#include "point.h"
#include "gaussian_mixture_factory.h"

namespace pclem {
    bool GaussianMixtureFactory::random_seeded = false;

    GaussianMixtureFactory::GaussianMixtureFactory() {
        if(!random_seeded) {
            srand(static_cast <unsigned> (time(0)));
        }
    }

    GaussianMixture GaussianMixtureFactory::from_pcl_corners(const PointCloud& pcl, double weight_of_parent_in_hierarchy) const {
        return from_pcl_corners(pcl.getBoundingBox(), weight_of_parent_in_hierarchy);
    }

    GaussianMixture GaussianMixtureFactory::from_pcl_corners(const DevicePointCloud& pcl, double weight_of_parent_in_hierarchy) const {
        return from_pcl_corners(pcl.getBoundingBox(), weight_of_parent_in_hierarchy);
    }

    GaussianMixture GaussianMixtureFactory::from_pcl_corners(const BoundingBox& bounding_box, double weight_of_parent_in_hierarchy) const {
        auto corners = bounding_box.corners();
        double initial_weight_of_gaussian = 1.0 / corners.size();

        Matrix33 sigma = covariance_from_pcl_corners(bounding_box);

        std::vector<WeightedGaussian> temp_gaussians;
        for(Point corner : corners) {
            WeightedGaussian gaussian(corner, sigma, initial_weight_of_gaussian, weight_of_parent_in_hierarchy);
            VLOG(4) << "Adding gaussian: " << gaussian;
            temp_gaussians.push_back(gaussian);
        }

        GaussianMixture mixture(temp_gaussians);
        return mixture;
    }

    // A heuristic to make the initial covariance matrix proportional to the size
    // of the bounding box of the point cloud.
    Matrix33 GaussianMixtureFactory::covariance_from_pcl_corners(const BoundingBox& bounding_box) const {
        Matrix33 m = Matrix33::zeros();

        Point min = bounding_box.getMin();
        Point max = bounding_box.getMax();

        double dx = max.x - min.x;
        double dy = max.y - min.y;
        double dz = max.z - min.z;

        m.set_element(0,0, (dx*dx/16));
        m.set_element(1,1, (dy*dy/16));
        m.set_element(2,2, (dz*dz/16));

        VLOG(1) << "Initial variances. X: " << m.get_element(0,0) << "Y: " << m.get_element(1,1) << "Z: " << m.get_element(2,2);

        return m;
    }

    GaussianMixture GaussianMixtureFactory::around_point(const Point& point, const Matrix33& cov,
                                                         int n_of_distributions, double delta, double weight_of_parent_in_hierarchy) const {
        VLOG(10) << "Creating gaussian mixture around point...";

        Matrix33 sigma;
        sigma.set_element(0,0,1.0);
        sigma.set_element(1,1,1.0);
        sigma.set_element(2,2,1.0);

        std::vector<WeightedGaussian> temp_gaussians;

        for(int i = 0; i < n_of_distributions; i++) {
            WeightedGaussian gaussian(
                Point(point.x + random_number(-delta, delta),
                      point.y + random_number(-delta, delta),
                      point.z + random_number(-delta, delta)),
                sigma,
                1.0 / n_of_distributions,
                weight_of_parent_in_hierarchy);

            temp_gaussians.push_back(gaussian);
        }

        GaussianMixture resulting_mixture(temp_gaussians);

        VLOG(10) << "Done creating gaussian mixture.";
        return resulting_mixture;
    }

    double GaussianMixtureFactory::random_number(double min, double max) {
        return min + static_cast<double> (rand()) / static_cast<double> (RAND_MAX/(max-min));
    }
}
