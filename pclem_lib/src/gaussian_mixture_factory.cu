#include "hip/hip_runtime.h"

#include <cstdlib>
#include <ctime>
#include <glog/logging.h>

#include "point.h"
#include "gaussian_mixture_factory.h"

namespace pclem {
    bool GaussianMixtureFactory::random_seeded = false;

    GaussianMixtureFactory::GaussianMixtureFactory() {
        if(!random_seeded) {
            srand(static_cast <unsigned> (time(0)));
        }
    }

    GaussianMixture GaussianMixtureFactory::from_pcl_corners(const PointCloud& pcl, double weight_of_parent_in_hierarchy) const {
        return from_pcl_corners(pcl.getBoundingBox(), weight_of_parent_in_hierarchy);
    }

    GaussianMixture GaussianMixtureFactory::from_pcl_corners(const DevicePointCloud& pcl, double weight_of_parent_in_hierarchy) const {
        return from_pcl_corners(pcl.getBoundingBox(), weight_of_parent_in_hierarchy);
    }

    GaussianMixture GaussianMixtureFactory::from_pcl_corners(const BoundingBox& bounding_box, double weight_of_parent_in_hierarchy) const {
        auto corners = bounding_box.corners();
        double initial_weight_of_gaussian = 1.0 / corners.size();

        std::vector<WeightedGaussian> temp_gaussians;
        for(Point corner : corners) {
            CovarianceMatrix sigma = CovarianceMatrix();
            sigma.set(0,0,10.0);
            sigma.set(1,1,10.0);
            sigma.set(2,2,10.0);

            WeightedGaussian gaussian(corner, sigma, initial_weight_of_gaussian, weight_of_parent_in_hierarchy);
            VLOG(4) << "Adding gaussian: " << gaussian;
            temp_gaussians.push_back(gaussian);
        }

        GaussianMixture mixture(temp_gaussians);
        return mixture;
    }

    GaussianMixture GaussianMixtureFactory::around_point(const Point& point, const CovarianceMatrix& cov,
                                                         int n_of_distributions, double delta, double weight_of_parent_in_hierarchy) const {
        VLOG(10) << "Creating gaussian mixture around point...";

        CovarianceMatrix sigma;
        sigma.set(0,0,1.0);
        sigma.set(1,1,1.0);
        sigma.set(2,2,1.0);

        std::vector<WeightedGaussian> temp_gaussians;

        for(int i = 0; i < n_of_distributions; i++) {
            WeightedGaussian gaussian(
                Point(point.x + random_number(-delta, delta),
                      point.y + random_number(-delta, delta),
                      point.z + random_number(-delta, delta)),
                sigma,
                1.0 / n_of_distributions,
                weight_of_parent_in_hierarchy);

            temp_gaussians.push_back(gaussian);
        }

        GaussianMixture resulting_mixture(temp_gaussians);

        VLOG(10) << "Done creating gaussian mixture.";
        return resulting_mixture;
    }

    double GaussianMixtureFactory::random_number(double min, double max) {
        return min + static_cast<double> (rand()) / static_cast<double> (RAND_MAX/(max-min));
    }
}
