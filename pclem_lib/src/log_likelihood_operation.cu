#include "hip/hip_runtime.h"

#include <glog/logging.h>
#include <thrust/transform_reduce.h>

#include "associated_point.cuh"
#include "log_likelihood_operation.h"

namespace pclem {
    LogLikelihoodOperation::LogLikelihoodOperation(const GaussianMixture& mixture) :
        mixture(mixture) {}

    double LogLikelihoodOperation::operator()(const DevicePointCloud::PointIterator& begin, const DevicePointCloud::PointIterator& end) const {
        double log_likelihood = 0.0;

        for(int i=0; i < AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE; i++) {
            WeightedGaussian distribution = mixture.get_gaussian(i);

            double likelihood_of_distribution = 0.0;
            if(distribution.get_weight() != 0.0) {
                likelihood_of_distribution = log_likelihood_of_distribution(begin, end,
                                                                            i, distribution);
            }

            VLOG(4) << "Mixture was: " << mixture;
            VLOG(4) << "Likelihood was: " << likelihood_of_distribution;

            log_likelihood += likelihood_of_distribution;
        }

        return log_likelihood;
    }

    struct log_likelihood_op : public thrust::unary_function<AssociatedPoint,double> {
        __const__ double log_pi_j;
        __const__ double base;
        __const__ DevicePoint mu;
        __const__ int index_of_distribution;
        double inv_of_cov[9];

        log_likelihood_op(int index_of_distribution, const WeightedGaussian& distribution) :
            log_pi_j(distribution.get_weight()),
            base(1.0 / sqrt(pow(2*M_PI, 3) * distribution.get_sigma().det())),
            mu(distribution.get_mu()),
            index_of_distribution(index_of_distribution),
            inv_of_cov {0.0} {

            VLOG(12) << "To inverse: " << distribution.get_sigma();
            std::array<double,9> inv_of_sigma = distribution.get_sigma().inverse();

            for(int i=0; i < 9; i++) {
                inv_of_cov[i] = inv_of_sigma[i];
            }
        }

        __host__ __device__
        double operator()(AssociatedPoint p) {
            if(p.associations[index_of_distribution] < 1e-30) {
                return 0.0;
            } else {
                return p.associations[index_of_distribution] * (log_pi_j + log(likelihood_of_point(p)));
            }
        }

    private:
        __host__ __device__
        double likelihood_of_point(DevicePoint p) {
            DevicePoint x_minus_mu = p - mu;

            double temp_product[3] = {x_minus_mu.z*inv_of_cov[6] + x_minus_mu.y*inv_of_cov[3] + x_minus_mu.x*inv_of_cov[0],
                                      x_minus_mu.z*inv_of_cov[7] + x_minus_mu.y*inv_of_cov[4] + x_minus_mu.x*inv_of_cov[1],
                                      x_minus_mu.z*inv_of_cov[8] + x_minus_mu.y*inv_of_cov[5] + x_minus_mu.x*inv_of_cov[2]};

            double scale_product = x_minus_mu.x * temp_product[0] +
                x_minus_mu.y * temp_product[1] +
                x_minus_mu.z * temp_product[2];

            return base * exp(-0.5 * scale_product);
        }
    };

    double LogLikelihoodOperation::log_likelihood_of_distribution(const DevicePointCloud::PointIterator& begin,
                                                                  const DevicePointCloud::PointIterator& end,
                                                                  int index_of_distribution, const WeightedGaussian& distribution) const {
        return thrust::transform_reduce(begin, end,
                                        log_likelihood_op(index_of_distribution, distribution),
                                        0.0, thrust::plus<double>());
    }
}
