#include "hip/hip_runtime.h"

#include <limits>

#include "bounding_box_creation_operation.h"

namespace pclem {
    struct min_op: public thrust::binary_function<DevicePoint,DevicePoint,DevicePoint> {
        __device__
        DevicePoint operator()(DevicePoint lhs, DevicePoint rhs) {
            return DevicePoint(thrust::min(lhs.x, rhs.x),
                               thrust::min(lhs.y, rhs.y),
                               thrust::min(lhs.z, rhs.z));
        }
    };

    struct max_op: public thrust::binary_function<DevicePoint,DevicePoint,DevicePoint> {
        __device__
        DevicePoint operator()(DevicePoint lhs, DevicePoint rhs) {
            return DevicePoint(thrust::max(lhs.x, rhs.x),
                               thrust::max(lhs.y, rhs.y),
                               thrust::max(lhs.z, rhs.z));
        }
    };

    BoundingBox BoundingBoxCreationOperation::operator()(const DevicePointCloud::PointIterator& begin,
                                                         const DevicePointCloud::PointIterator& end) {
        min_op min_function;
        max_op max_function;
        double pos_inf = std::numeric_limits<double>::infinity();
        double neg_inf = -1*std::numeric_limits<double>::infinity();

        DevicePoint min = thrust::reduce(begin, end, DevicePoint(pos_inf, pos_inf, pos_inf), min_function);
        DevicePoint max = thrust::reduce(begin, end, DevicePoint(neg_inf, neg_inf, neg_inf), max_function);

        Point host_min = min.to_host();
        Point host_max = max.to_host();

        BoundingBox bounding_box(host_min, host_max);

        return bounding_box;
    }
}
