#include "hip/hip_runtime.h"

#include "bounding_box_creation_operation.h"

namespace pclem {
    struct min_op: public thrust::binary_function<DevicePoint,DevicePoint,DevicePoint> {
        __device__
        DevicePoint operator()(DevicePoint lhs, DevicePoint rhs) {
            return DevicePoint(thrust::min(lhs.x, rhs.x),
                               thrust::min(lhs.y, rhs.y),
                               thrust::min(lhs.z, rhs.z));
        }
    };

    struct max_op: public thrust::binary_function<DevicePoint,DevicePoint,DevicePoint> {
        __device__
        DevicePoint operator()(DevicePoint lhs, DevicePoint rhs) {
            return DevicePoint(thrust::max(lhs.x, rhs.x),
                               thrust::max(lhs.y, rhs.y),
                               thrust::max(lhs.z, rhs.z));
        }
    };

    BoundingBox BoundingBoxCreationOperation::operator()(const DevicePointCloud::PointIterator& begin,
                                                         const DevicePointCloud::PointIterator& end) {
        min_op min_function;
        max_op max_function;

        DevicePoint min = thrust::reduce(begin, end, DevicePoint(0.0,0.0,0.0), min_function);
        DevicePoint max = thrust::reduce(begin, end, DevicePoint(0.0,0.0,0.0), max_function);

        Point host_min = min.to_host();
        Point host_max = max.to_host();

        BoundingBox bounding_box(host_min, host_max);

        return bounding_box;
    }
}
