#include "hip/hip_runtime.h"

#include "boundingbox.h"

namespace pclem {
    BoundingBox::BoundingBox() :
        min(Point(0.0,0.0,0.0)), max(Point(0.0,0.0,0.0)) {
    }

    BoundingBox::BoundingBox(Point min, Point max) :
        min(min), max(max) {
    }

    BoundingBox::BoundingBox(const BoundingBox& other) :
        min(other.min), max(other.max) {}

    void BoundingBox::setMin(Point pmin) {
        min = pmin;
    }

    void BoundingBox::setMax(Point pmax) {
        max = pmax;
    }

    std::vector<Point> BoundingBox::corners() const {
        auto corners = std::vector<Point>();

        corners.push_back(Point(max.x, max.y, max.z));
        corners.push_back(Point(max.x, max.y, min.z));
        corners.push_back(Point(max.x, min.y, max.z));
        corners.push_back(Point(min.x, max.y, max.z));
        corners.push_back(Point(max.x, min.y, min.z));
        corners.push_back(Point(min.x, min.y, max.z));
        corners.push_back(Point(min.x, max.y, min.z));
        corners.push_back(Point(min.x, min.y, min.z));

        return corners;
    }


    BoundingBox& BoundingBox::operator=(const BoundingBox& other) {
        min = other.min;
        max = other.max;

        return *this;
    }

    double BoundingBox::volume() const {
        double deltax = max.x - min.x;
        double deltay = max.y - min.y;
        double deltaz = max.z - min.z;

        return deltax * deltay * deltaz;
    }

    Point BoundingBox::getMin() const {
        return min;
    }
    Point BoundingBox::getMax() const {
        return max;
    }
}
