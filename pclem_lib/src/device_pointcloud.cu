#include "hip/hip_runtime.h"

#include <math.h>
#include <cstdio>
#include <chrono>
#include <limits>
#include <glog/logging.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#include <armadillo>

#include "device_pointcloud.h"
#include "raw_covariance_matrix.cuh"
#include "gaussian_mixture_factory.h"
#include "device_hierarchical_gaussian_mixture.h"
#include "em_algorithm.h"

#include "association_computing_operation.h"
#include "point_normalization_operation.h"
#include "log_likelihood_operation.h"

namespace pclem {
    DevicePointCloud::DevicePointCloud() :
        ptr_to_points(new thrust::device_vector<AssociatedPoint>()),
        pts_begin(ptr_to_points->begin()),
        pts_end(ptr_to_points->end()),
        boundingBox() {}

    DevicePointCloud::DevicePointCloud(const DevicePointCloud& other) :
        ptr_to_points(other.ptr_to_points),
        pts_begin(other.pts_begin),
        pts_end(other.pts_end),
        boundingBox(other.boundingBox){}

    BoundingBox DevicePointCloud::getBoundingBox() const {
        return BoundingBox(boundingBox);
    }

    struct min_op: public thrust::binary_function<DevicePoint,DevicePoint,DevicePoint> {
        __device__
        DevicePoint operator()(DevicePoint lhs, DevicePoint rhs) {
            return DevicePoint(thrust::min(lhs.x, rhs.x),
                               thrust::min(lhs.y, rhs.y),
                               thrust::min(lhs.z, rhs.z));
        }
    };

    struct max_op: public thrust::binary_function<DevicePoint,DevicePoint,DevicePoint> {
        __device__
        DevicePoint operator()(DevicePoint lhs, DevicePoint rhs) {
            return DevicePoint(thrust::max(lhs.x, rhs.x),
                               thrust::max(lhs.y, rhs.y),
                               thrust::max(lhs.z, rhs.z));
        }
    };

    void DevicePointCloud::updateBoundingBox(){
        min_op min_function;
        max_op max_function;

        DevicePoint min = thrust::reduce(pts_begin, pts_end, DevicePoint(0.0,0.0,0.0), min_function);
        DevicePoint max = thrust::reduce(pts_begin, pts_end, DevicePoint(0.0,0.0,0.0), max_function);

        Point host_min = min.to_host();
        Point host_max = max.to_host();

        boundingBox.setMin(host_min);
        boundingBox.setMax(host_max);
    }

    int DevicePointCloud::get_n_points() const {
        return pts_end - pts_begin;
    }

    void DevicePointCloud::set_points(const std::shared_ptr<thrust::device_vector<AssociatedPoint>>& points) {
        VLOG(10) << "Setting new data source...";

        ptr_to_points = points;
        pts_begin = points->begin();
        pts_end = points->end();

        updateBoundingBox();

        VLOG(10) << "Done setting data source.";
    }

    void DevicePointCloud::set_points(const std::shared_ptr<thrust::device_vector<AssociatedPoint>>& points,
                                      const PointIterator& begin,
                                      const PointIterator& end) {
        ptr_to_points = points;
        pts_begin = begin;
        pts_end = end;
    }

    void DevicePointCloud::compute_associations(const GaussianMixture& mixture) {
        VLOG(10) << "Computing point/distribution associations...";

        AssociationComputingOperation op(mixture);
        execute_pointcloud_operation(op);

        VLOG(10) << "Done computing point/distribution associations.";
    }

    void DevicePointCloud::normalize_associations() {
        execute_pointcloud_operation<void>(PointNormalizationOperation());
    }

    // This functor stores the sums of gammas in an AssociatedPoint.
    struct sums_of_gammas_op : public thrust::binary_function<AssociatedPoint, AssociatedPoint, AssociatedPoint> {

        __host__ __device__
        AssociatedPoint operator()(AssociatedPoint lhs, AssociatedPoint rhs) {
            AssociatedPoint sum;
            for(int i=0; i < AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE; i++) {
                sum.likelihoods[i] = lhs.likelihoods[i] + rhs.likelihoods[i];
            }
            return sum;
        }
    };

    GaussianMixture DevicePointCloud::create_mixture() const {
        // We store the sum of gammas of every distribution in an empty, meaningless AssociatedPoint.
        AssociatedPoint sums;
        sums = thrust::reduce(pts_begin, pts_end, AssociatedPoint(), sums_of_gammas_op());

        std::vector<WeightedGaussian> gaussians;

        for(int i=0; i < AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE; i++) {
            gaussians.push_back(create_distribution_of_mixture(i, sums.likelihoods[i]));
        }

        return GaussianMixture(gaussians);
    }

    struct weight_point_op : public thrust::unary_function<AssociatedPoint,DevicePoint> {
        int index_of_distribution;

        weight_point_op(int index_of_distribution) : index_of_distribution(index_of_distribution) {}

        __host__ __device__
        DevicePoint operator()(AssociatedPoint p) {
            return DevicePoint(p.x * p.likelihoods[index_of_distribution],
                               p.y * p.likelihoods[index_of_distribution],
                               p.z * p.likelihoods[index_of_distribution]);
        }
    };

    struct sum_of_points_op : public thrust::binary_function<DevicePoint,DevicePoint,DevicePoint> {
        __host__ __device__
        DevicePoint operator()(DevicePoint lhs, DevicePoint rhs) {
            return lhs + rhs;
        }
    };

    WeightedGaussian DevicePointCloud::create_distribution_of_mixture(int index_of_distribution, double sum_of_gammas) const {
        VLOG(10) << "Creating distribution " << index_of_distribution << " of mixture...";

        DevicePoint new_mu = thrust::transform_reduce(pts_begin, pts_end,
                                                      weight_point_op(index_of_distribution),
                                                      DevicePoint(0.0, 0.0, 0.0),
                                                      sum_of_points_op());

        new_mu = DevicePoint(new_mu.x / sum_of_gammas,
                             new_mu.y / sum_of_gammas,
                             new_mu.z / sum_of_gammas);

        VLOG(1) << "New mu: " << new_mu;

        CovarianceMatrix new_sigma = compute_sigma(index_of_distribution, new_mu.to_host(), sum_of_gammas);

        double new_weight = sum_of_gammas / get_n_points();

        VLOG(10) << "Done creating distribution " << index_of_distribution << " of mixture.";

        Point pub_new_mu = new_mu.to_host();
        WeightedGaussian to_return = WeightedGaussian(pub_new_mu, new_sigma, new_weight);

        return to_return;
    }

    struct point_to_cov_op : public thrust::unary_function<AssociatedPoint, RawCovarianceMatrix> {
        int index_of_distribution;

        point_to_cov_op(int index_of_distribution) : index_of_distribution(index_of_distribution) {}

        __host__ __device__
        RawCovarianceMatrix operator()(AssociatedPoint p) {
            RawCovarianceMatrix r;
            double gamma = p.likelihoods[index_of_distribution];

            r.v00 = p.x * p.x * gamma;
            r.v11 = p.y * p.y * gamma;
            r.v22 = p.z * p.z * gamma;

            r.v10 = r.v01 = p.x * p.y * gamma;
            r.v20 = r.v02 = p.x * p.z * gamma;
            r.v21 = r.v12 = p.y * p.z * gamma;

            return r;
        }
    };

    struct sum_of_cov_op : public thrust::binary_function<RawCovarianceMatrix, RawCovarianceMatrix, RawCovarianceMatrix> {
        __host__ __device__
        RawCovarianceMatrix operator()(RawCovarianceMatrix lhs, RawCovarianceMatrix rhs) {
            return lhs + rhs;
        }
    };

    CovarianceMatrix DevicePointCloud::compute_sigma(int index_of_distribution, const Point& mu, double sum_of_gammas) const {
        RawCovarianceMatrix init = RawCovarianceMatrix::zeros();

        RawCovarianceMatrix sigma = thrust::transform_reduce(pts_begin, pts_end,
                                                             point_to_cov_op(index_of_distribution),
                                                             init, sum_of_cov_op());

        RawCovarianceMatrix base_sigma;

        base_sigma.v00 = mu.x * mu.x;
        base_sigma.v11 = mu.y * mu.y;
        base_sigma.v22 = mu.z * mu.z;

        base_sigma.v01 = base_sigma.v10 = mu.x * mu.y;
        base_sigma.v02 = base_sigma.v20 = mu.x * mu.z;
        base_sigma.v21 = base_sigma.v12 = mu.z * mu.y;

        sigma = (sigma / sum_of_gammas) - base_sigma;
        return sigma.to_host();
    }

    double DevicePointCloud::log_likelihood_of_mixture(const GaussianMixture& mixture) const {
        LogLikelihoodOperation op(mixture);
        return execute_pointcloud_operation<double>(op);
    }

    std::vector<Point> DevicePointCloud::copy_of_points() const {
        std::vector<Point> copy;

        for(auto it = pts_begin; it != pts_end; it++) {
            AssociatedPoint device_point(*it);
            copy.push_back(device_point.to_host());
        }

        return copy;
    }

    HierarchicalGaussianMixture DevicePointCloud::create_hgmm() {
        VLOG(10) << "Creating hgmm...";
        GaussianMixtureFactory gmm_factory;

        PointCloud vanilla_pcl(new DevicePointCloud(*this));
        EmAlgorithm em_algorithm = EmAlgorithm::from_pcl(vanilla_pcl);
        em_algorithm.run(0.0001);

        std::cout << em_algorithm;

        std::shared_ptr<DeviceHierarchicalGaussianMixture> hierarchical_mixture(new DeviceHierarchicalGaussianMixture(*this, gmm_factory.from_pcl_corners(*this)));
        hierarchical_mixture->create_children();

        VLOG(10) << "Done creating hgmm.";
        return HierarchicalGaussianMixture(hierarchical_mixture);
    }

    DevicePointCloud::PointIterator DevicePointCloud::begin() {
        return pts_begin;
    }

    DevicePointCloud::PointIterator DevicePointCloud::end() {
        return pts_end;
    }

    std::shared_ptr<thrust::device_vector<AssociatedPoint>> DevicePointCloud::get_data() const {
        return ptr_to_points;
    }

    template<typename T>
    T DevicePointCloud::execute_pointcloud_operation(DevicePointCloud::PointCloudOperation<T> op) const {
        return op(pts_begin, pts_end);
    }
}
