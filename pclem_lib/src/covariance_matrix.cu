#include "hip/hip_runtime.h"

#include <glog/logging.h>
#include <armadillo>
#include "covariance_matrix.h"

namespace pclem {
    CovarianceMatrix::CovarianceMatrix() : values {{1.0,0.0,0.0,0.0,1.0,0.0,0.0,0.0,1.0}} {}

    CovarianceMatrix::CovarianceMatrix(const CovarianceMatrix& other) {
        values = other.values;
    }

    CovarianceMatrix::CovarianceMatrix(std::array<double,9> _values) {
        for(int i = 0; i < 9; i++) {
            values[i] = _values[i];
        }
    }

    CovarianceMatrix CovarianceMatrix::zeros() {
        std::array<double,9> zeros = {0.0};
        return CovarianceMatrix(zeros);
    }

    void CovarianceMatrix::operator+=(const CovarianceMatrix& rhs) {
        for(int i = 0; i < 9; i++) {
            values[i] += rhs.values[i];
        }
    }

    double CovarianceMatrix::get(int i, int j) const {
        return values[i*3 + j];
    }

    void CovarianceMatrix::set(int i, int j, double new_value) {
        values[i*3 + j] = new_value;
    }

    double CovarianceMatrix::det() const {
        return get(0,0) * (get(1,1) * get(2,2) - get(2, 1) * get(1,2)) -
            get(0,1) * (get(1,0) * get(2,2) - get(1,2) * get(2,0)) +
            get(0,2) * (get(1,0) * get(2,1) + get(2,0) * get(1,1));
    }

    std::array<double,9> CovarianceMatrix::as_array() const {
        auto m = values;
        return m;
    }

    std::array<double,9> CovarianceMatrix::inverse() const {
        VLOG(11) << "Inverting matrix...";

        arma::mat33 arma_cov_mat(values.data());
        arma::mat33 arma_inv_of_cov = arma::pinv(arma_cov_mat + arma::eye(3,3));

        VLOG(11) << "PseudoInverse error " << arma::norm(arma_cov_mat*arma_inv_of_cov - arma::eye(3,3));

        std::array<double,9> inverse;
        for(auto i = 0; i < 3; i++) {
            for(auto j = 0; j < 3; j++) {
                inverse[i*3 + j] = arma_inv_of_cov(i,j);
            }
        }

        VLOG(11) << "Done inverting matrix.";
        return inverse;
    }

    std::pair<Vector3, Matrix33> CovarianceMatrix::eigen_decomposition() const {
        VLOG(10) << "Extracting eigenvalues...";

        arma::mat33 arma_cov_mat(values.data());
        arma::vec arma_eigvals;
        arma::mat arma_eigvecs;

        if(!arma::eig_sym(arma_eigvals, arma_eigvecs, arma_cov_mat)) {
            std::cout << "DECOMPOTISION FAILDE";
            LOG(WARNING) << "Eigenvalues decomposition failed.";
        }

        Vector3 eigvals;
        std::array<double,9> eigvecs_values;
        for(int i=0; i < 3; i++) {
            eigvals[i] = arma_eigvals[i];
            std::cout << "Eigval " << eigvals[i] << std::endl;

            for(int j=0; j < 3; j++) {
                std::cout << arma_eigvecs(i,j);
                eigvecs_values[i*3 + j] = arma_eigvecs(i,j);
            }
        }

        return std::make_pair(eigvals, Matrix33(eigvecs_values));
    }

    std::ostream& operator<<(std::ostream& os, const CovarianceMatrix& m) {
        os << "[";
        for(auto value : m.as_array()) {
            os << value << ",";
        }
        os << "]";

        return os;
    }

    std::ofstream& operator<<(std::ofstream& ofs, const CovarianceMatrix& m) {
        std::array<double,9> array_of_matrix = m.as_array();
        for(int i=0; i < 8; i++) {
            ofs << array_of_matrix[i] << ",";
        }
        ofs << array_of_matrix[8];

        return ofs;
    }

    CovarianceMatrix CovarianceMatrix::identity() {
        return CovarianceMatrix({1.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0, 1.0});
    }
}
