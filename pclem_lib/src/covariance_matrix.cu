#include "hip/hip_runtime.h"

#include <glog/logging.h>
#include <armadillo>
#include "covariance_matrix.h"

namespace pclem {
    CovarianceMatrix::CovarianceMatrix() : values {{1.0,0.0,0.0,0.0,1.0,0.0,0.0,0.0,1.0}} {}

    CovarianceMatrix::CovarianceMatrix(const CovarianceMatrix& other) {
        values = other.values;
    }

    CovarianceMatrix::CovarianceMatrix(std::array<double,9> _values) {
        for(int i = 0; i < 9; i++) {
            values[i] = _values[i];
        }
    }

    CovarianceMatrix CovarianceMatrix::zeros() {
        std::array<double,9> zeros = {0.0};
        return CovarianceMatrix(zeros);
    }

    void CovarianceMatrix::operator+=(const CovarianceMatrix& rhs) {
        for(int i = 0; i < 9; i++) {
            values[i] += rhs.values[i];
        }
    }

    double CovarianceMatrix::get(int i, int j) const {
        return values[i*3 + j];
    }

    void CovarianceMatrix::set(int i, int j, double new_value) {
        values[i*3 + j] = new_value;
    }

    double CovarianceMatrix::det() const {
        return get(0,0) * (get(1,1) * get(2,2) - get(2, 1) * get(1,2)) -
            get(0,1) * (get(1,0) * get(2,2) - get(1,2) * get(2,0)) +
            get(0,2) * (get(1,0) * get(2,1) + get(2,0) * get(1,1));
    }

    std::array<double,9> CovarianceMatrix::as_array() const {
        auto m = values;
        return m;
    }

    std::array<double,9> CovarianceMatrix::inverse() const {
        VLOG(11) << "Inverting matrix...";

        arma::mat33 arma_cov_mat(values.data());
        arma::mat33 arma_inv_of_cov = arma::inv(arma_cov_mat);

        std::array<double,9> inverse;
        for(auto i = 0; i < 3; i++) {
            for(auto j = 0; j < 3; j++) {
                inverse[i*3 + j] = arma_inv_of_cov(i,j);
            }
        }

        VLOG(11) << "Done inverting matrix.";
        return inverse;
    }

    std::pair<Vector3, std::array<Vector3,3>> CovarianceMatrix::eigenvalues() const {
        VLOG(10) << "Extracting eigenvalues...";

        arma::mat33 arma_cov_mat(values.data());
        arma::cx_vec arma_eigvals;
        arma::cx_mat arma_eigvecs;

        arma::eig_gen(arma_eigvals, arma_eigvecs, arma_cov_mat);

        Vector3 eigvals;
        std::array<Vector3, 3> eigvecs;

        for(int i=0; i < 3; i++) {
            eigvals[i] = arma_eigvals[i].real();

            for(int j=0; j < 3; j++) {
                eigvecs[i][j] = arma_eigvecs(j,i).real();
            }

            if(std::abs(arma_eigvals[i].imag()) > 1e-300) {
                LOG(WARNING) << "Imaginary part was found during eigenvalue extraction";
            }
        }

        return std::make_pair(eigvals, eigvecs);
    }

    std::ostream& operator<<(std::ostream& os, const CovarianceMatrix& m) {
        os << "[";
        for(auto value : m.as_array()) {
            os << value << ",";
        }
        os << "]";

        return os;
    }
}
