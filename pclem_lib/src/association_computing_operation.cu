#include "hip/hip_runtime.h"

#include <glog/logging.h>

#include "association_computing_operation.h"

namespace pclem {
    
    AssociationComputingOperation::AssociationComputingOperation(const GaussianMixture& mixture) :
        mixture(mixture) {}

    void AssociationComputingOperation::operator()(const DevicePointCloud::PointIterator& begin, const DevicePointCloud::PointIterator& end) {
        VLOG(10) << "Computing point/distribution associations...";

        for(int i = 0; i < mixture.n_gaussians(); i++) {
            WeightedGaussian distribution = mixture.get_gaussian(i);

            if(distribution.get_weight() == 0.0) {
                VLOG(1) << "Found dropped distribution.";
                execute_no_association_op(begin, end, i);
            } else {
                compute_associations_of_distribution(begin, end, i, mixture.get_gaussian(i));
            }

        }

        VLOG(10) << "Done computing point/distribution associations.";
    }

    struct gaussian_op : public thrust::unary_function<AssociatedPoint,AssociatedPoint> {
    public:
        gaussian_op(int index_of_distribution, const WeightedGaussian& distribution) :
            index_of_distribution(index_of_distribution),
            mu(distribution.get_mu()),
            base(distribution.get_weight() / sqrt(pow(2*M_PI,3) * distribution.get_sigma().det())),
            inv_of_covariance {0.0} {
            std::array<double,9> computed_inv = distribution.get_sigma().inverse();

            for (int i = 0; i < 9; i++) {
                inv_of_covariance[i] = computed_inv[i];
            }
        }

        __host__ __device__
        AssociatedPoint operator()(AssociatedPoint p) {
            p.likelihoods[index_of_distribution] = likelihood_of_point(p);

            if(p.likelihoods[index_of_distribution] > p.likelihoods[p.best_distribution]) {
                p.best_distribution = index_of_distribution;
            }

            return p;
        }

    private:
        __const__ int index_of_distribution;
        __const__ DevicePoint mu;
        __const__ double base;
        double inv_of_covariance[9];

        __host__ __device__
        double likelihood_of_point(DevicePoint p) {
            DevicePoint x_minus_mu = p - mu;

            double temp_product[3] = {x_minus_mu.z*inv_of_covariance[6] + x_minus_mu.y*inv_of_covariance[3] + x_minus_mu.x*inv_of_covariance[0],
                                      x_minus_mu.z*inv_of_covariance[7] + x_minus_mu.y*inv_of_covariance[4] + x_minus_mu.x*inv_of_covariance[1],
                                      x_minus_mu.z*inv_of_covariance[8] + x_minus_mu.y*inv_of_covariance[5] + x_minus_mu.x*inv_of_covariance[2]};

            double scale_product = x_minus_mu.x * temp_product[0] +
                x_minus_mu.y * temp_product[1] +
                x_minus_mu.z * temp_product[2];

            return base * exp(-0.5 * scale_product);
        }
    };

    void AssociationComputingOperation::compute_associations_of_distribution(const DevicePointCloud::PointIterator& begin,
                                                                             const DevicePointCloud::PointIterator& end,
                                                                             int index_of_distribution,
                                                                             const WeightedGaussian& distribution) {
        gaussian_op op(index_of_distribution, distribution);

        thrust::transform(begin, end, begin, op);
    }

    struct no_association_op : public thrust::unary_function<AssociatedPoint,AssociatedPoint> {
        int index_of_distribution;

        no_association_op(int index_of_distribution) :
            index_of_distribution(index_of_distribution) {}

        __host__ __device__
        AssociatedPoint operator()(AssociatedPoint p) {
            p.likelihoods[index_of_distribution] = 0.0;
            return p;
        }
    };

    void AssociationComputingOperation::execute_no_association_op(const DevicePointCloud::PointIterator& begin,
                                                                  const DevicePointCloud::PointIterator& end,
                                                                  int index_of_distribution) const {
        thrust::transform(begin, end, begin, no_association_op(index_of_distribution));
    }
}
