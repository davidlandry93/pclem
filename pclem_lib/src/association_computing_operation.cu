#include "hip/hip_runtime.h"

#include <glog/logging.h>

#include <thrust/transform.h>

#include "association_computing_operation.h"

namespace pclem {
    
    AssociationComputingOperation::AssociationComputingOperation(const GaussianMixture& mixture, const double& volume) :
        mixture(mixture), volume_of_pcl(volume) {}

    void AssociationComputingOperation::operator()(const DevicePointCloud::PointIterator& begin, const DevicePointCloud::PointIterator& end) {
        VLOG(10) << "Computing point/distribution associations...";

        for(int i = 0; i < AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE; i++) {
            select_and_execute_op(begin, end, i);
        }

        VLOG(10) << "Done computing point/distribution associations.";
    }

    struct gaussian_op : public thrust::unary_function<AssociatedPoint,AssociatedPoint> {
    public:
        gaussian_op(int index_of_distribution, const WeightedGaussian& distribution) :
            index_of_distribution(index_of_distribution),
            mu(distribution.get_mu()),
            base(distribution.get_weight() / sqrt(pow(2*M_PI,3) * distribution.get_sigma().det())),
            inv_of_covariance {0.0} {
            std::array<double,9> computed_inv = distribution.get_sigma().inverse();

            for (int i = 0; i < 9; i++) {
                inv_of_covariance[i] = computed_inv[i];
            }
        }

        __host__ __device__
        AssociatedPoint operator()(AssociatedPoint p) {
            p.associations[index_of_distribution] = likelihood_of_point(p);

            if(p.associations[index_of_distribution] > p.associations[p.best_distribution]) {
                p.best_distribution = index_of_distribution;
            }

            return p;
        }

    private:
        __const__ int index_of_distribution;
        __const__ DevicePoint mu;
        __const__ double base;
        double inv_of_covariance[9];

        __host__ __device__
        double likelihood_of_point(DevicePoint p) {
            DevicePoint x_minus_mu = p - mu;

            double temp_product[3] = {x_minus_mu.z*inv_of_covariance[6] + x_minus_mu.y*inv_of_covariance[3] + x_minus_mu.x*inv_of_covariance[0],
                                      x_minus_mu.z*inv_of_covariance[7] + x_minus_mu.y*inv_of_covariance[4] + x_minus_mu.x*inv_of_covariance[1],
                                      x_minus_mu.z*inv_of_covariance[8] + x_minus_mu.y*inv_of_covariance[5] + x_minus_mu.x*inv_of_covariance[2]};

            double scale_product = x_minus_mu.x * temp_product[0] +
                x_minus_mu.y * temp_product[1] +
                x_minus_mu.z * temp_product[2];

            return base * exp(-0.5 * scale_product);
        }
    };

    struct fixed_association_op : public thrust::unary_function<AssociatedPoint,AssociatedPoint> {
        int index_of_distribution;
        double association_value;

        fixed_association_op(int index_of_distribution, double association_value) :
            index_of_distribution(index_of_distribution),
            association_value(association_value) {}

        __host__ __device__
        AssociatedPoint operator()(AssociatedPoint p) {
            p.associations[index_of_distribution] = association_value;
            return p;
        }
    };

    void AssociationComputingOperation::select_and_execute_op(const DevicePointCloud::PointIterator& begin, const DevicePointCloud::PointIterator& end, const int& i) {
        if(mixture.get_gaussian(i).get_weight() == 0.0) {
            thrust::transform(begin, end, begin, fixed_association_op(i, 0.0));
        } else {
            thrust::transform(begin, end, begin, gaussian_op(i, mixture.get_gaussian(i)));
        }
    }
}
