#include "hip/hip_runtime.h"

#include <cmath>

#include "ellipsoid.h"
#include "weighted_gaussian.h"

namespace pclem {
    WeightedGaussian::WeightedGaussian() :mu(), sigma(), weight_in_mixture(0.0), weight_in_hierarchy_of_parent(0.0) {}

    WeightedGaussian::WeightedGaussian(const Point& mu, const Matrix33& sigma, double weight_in_mixture) :
        WeightedGaussian(mu, sigma, weight_in_mixture, 1.0) {}

    WeightedGaussian::WeightedGaussian(const Point& mu, const Matrix33& sigma, double weight_in_mixture, double weight_in_hierarchy_of_parent) :
        mu(mu), sigma(sigma), weight_in_mixture(weight_in_mixture), weight_in_hierarchy_of_parent(weight_in_hierarchy_of_parent) {}

    WeightedGaussian::WeightedGaussian(const WeightedGaussian& other) :
        mu(other.mu), sigma(other.sigma), weight_in_mixture(other.weight_in_mixture),
        weight_in_hierarchy_of_parent(other.weight_in_hierarchy_of_parent) {}

    Matrix33 WeightedGaussian::get_sigma() const {
        Matrix33 m(sigma);
        return m;
    }

    Point WeightedGaussian::get_mu() const {
        Point p(mu);
        return p;
    }

    double WeightedGaussian::get_weight() const {
        return weight_in_mixture;
    }

    double WeightedGaussian::weight_in_hierarchy() const {
        return weight_in_mixture * weight_in_hierarchy_of_parent;
    }

    void WeightedGaussian::insert_into_visualization(Visualization& vis) const {
        Matrix33 cov = get_sigma();

        auto eigen_result = cov.eigen_decomposition();
        Vector3 eigenvalues = eigen_result.first;
        Matrix33 eigenvectors = eigen_result.second;

        Vector3 position(get_mu().x, get_mu().y, get_mu().z);
        Ellipsoid ellipsoid(std::sqrt(eigenvalues[0]), std::sqrt(eigenvalues[1]), std::sqrt(eigenvalues[2]), position, eigenvectors, weight_in_hierarchy());

        vis.insert_ellipsoid(ellipsoid);
    }

    std::ostream &operator<<(std::ostream &os, WeightedGaussian const &g) {
        os << " WEIGHT: " << g.get_weight() <<
            " W IN HIERARCHY: " << g.weight_in_hierarchy() <<
            " MU: " << g.get_mu() <<
            " SIGMA: " << g.get_sigma();
        return os;
    }
}
