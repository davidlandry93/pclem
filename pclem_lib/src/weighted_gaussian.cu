#include "hip/hip_runtime.h"

#include <cmath>

#include "ellipsoid.h"
#include "weighted_gaussian.h"

namespace pclem {
    WeightedGaussian::WeightedGaussian() :mu(), sigma(), weight(0) {}

    WeightedGaussian::WeightedGaussian(const Point& mu, const CovarianceMatrix& sigma, double weight) :
        mu(mu), sigma(sigma), weight(weight) {}

    WeightedGaussian::WeightedGaussian(const WeightedGaussian& other) :
        mu(other.mu), sigma(other.sigma), weight(other.weight) {}

    CovarianceMatrix WeightedGaussian::get_sigma() const {
        CovarianceMatrix m(sigma);
        return m;
    }

    Point WeightedGaussian::get_mu() const {
        Point p(mu);
        return p;
    }

    double WeightedGaussian::get_weight() const {
        return weight;
    }

    void WeightedGaussian::insert_into_visualization(Visualization& vis) const {
        CovarianceMatrix cov = get_sigma();

        auto eigen_result = cov.svd_decomposition();
        Vector3 eigenvalues = eigen_result.first;
        Matrix33 eigenvectors = eigen_result.second;

        std::cout << "Eigenvalues: " << eigenvalues[0] << eigenvalues[1] << eigenvalues[2] << std::endl;

        Vector3 position(get_mu().x, get_mu().y, get_mu().z);
        Ellipsoid ellipsoid(std::sqrt(eigenvalues[0]), std::sqrt(eigenvalues[1]), std::sqrt(eigenvalues[2]), position, eigenvectors, get_weight());

        std::cout << ellipsoid;

        vis.insert_ellipsoid(ellipsoid);
    }

    std::ostream &operator<<(std::ostream &os, WeightedGaussian const &g) {
        os << "MU: " << g.get_mu() <<
            " WEIGHT: " << g.get_weight() <<
            " SIGMA: " << g.get_sigma();
        return os;
    }
}
