#include "hip/hip_runtime.h"

#include <memory>
#include <glog/logging.h>
#include <thrust/partition.h>
#include <thrust/find.h>

#include "em_algorithm.h"
#include "associated_point.cuh"
#include "device_hierarchical_gaussian_mixture.h"
#include "pointcloud.h"
#include "gaussian_mixture_factory.h"
#include "sort_by_best_association_operation.h"

namespace pclem {
    DeviceHierarchicalGaussianMixture::DeviceHierarchicalGaussianMixture(const DevicePointCloud& pcl, const GaussianMixture& mixture)
        : pcl(pcl), mixture(mixture), children() {}

    void DeviceHierarchicalGaussianMixture::create_children(std::vector<DeviceHierarchicalGaussianMixture*>& to_expand) {
        VLOG(10) << "Creating children of hierarchical gaussian mixture...";
        SortByBestAssociationOperation op;

        std::vector<DevicePointCloud::PointIterator> boundaries = pcl.execute_pointcloud_operation(op);

        //children.push_back(create_one_child(sub_pcl_begin, first_unsorted, mixture.get_gaussian(i)));

        VLOG(10) << "Done creating children";
    }

    void DeviceHierarchicalGaussianMixture::expand_n_levels(int n_levels) {
        VLOG(10) << "Expanding hierarchy of n levels...";

        run_em();
        std::cout << "Done with em";

        int n_nodes_to_create = std::pow(AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE, n_levels);
        std::vector<DeviceHierarchicalGaussianMixture*> to_expand;
        std::cout << "Adding root to queue";
        to_expand.push_back(this);

        while(n_nodes_to_create > 0) {

            n_nodes_to_create -= AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE;
        }

        VLOG(10) << "Done expanding hierarchy of n levels.";
    }

    void DeviceHierarchicalGaussianMixture::run_em() {
        auto ptr = std::shared_ptr<DevicePointCloud>(new DevicePointCloud(pcl));
        PointCloud vanilla_pcl(ptr);

        auto em = EmAlgorithm::from_pcl(vanilla_pcl);
        em.run(EM_CONVERGENCE_THRESHOLD);
    }

    DeviceHierarchicalGaussianMixture DeviceHierarchicalGaussianMixture::create_one_child(const DevicePointCloud::PointIterator& begin,
                                                                                          const DevicePointCloud::PointIterator& end,
                                                                                          const WeightedGaussian& parent_distribution) const {
        GaussianMixtureFactory gmm_factory;
        DevicePointCloud child_pcl;

        VLOG(1) << "N of points in child :" << end - begin;

        child_pcl.set_points(pcl.get_data(), begin, end);
        auto child_mixture = gmm_factory.around_point(parent_distribution.get_mu(),
                                                      parent_distribution.get_sigma(),
                                                      UNIFORM_DISTRIBUTION_SIZE,
                                                      AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE);

        return DeviceHierarchicalGaussianMixture(child_pcl, child_mixture);
    }
}
