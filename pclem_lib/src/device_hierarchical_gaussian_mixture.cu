#include "hip/hip_runtime.h"

#include <memory>
#include <glog/logging.h>
#include <thrust/partition.h>
#include <thrust/find.h>
#include <deque>

#include "em_algorithm.h"
#include "associated_point.cuh"
#include "device_hierarchical_gaussian_mixture.h"
#include "pointcloud.h"
#include "gaussian_mixture_factory.h"
#include "sort_by_best_association_operation.h"

namespace pclem {
    DeviceHierarchicalGaussianMixture::DeviceHierarchicalGaussianMixture(const DevicePointCloud& pcl, const GaussianMixture& mixture)
        : pcl(pcl), mixture(mixture), children() {}

    void DeviceHierarchicalGaussianMixture::expand_n_levels(int n_levels) {
        VLOG(10) << "Expanding hierarchy of n levels...";

        run_em();

        int n_nodes_to_create = std::pow(AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE, n_levels);
        std::deque<DeviceHierarchicalGaussianMixture*> to_expand;

        to_expand.push_back(this);
        while(n_nodes_to_create > 0) {
            DeviceHierarchicalGaussianMixture* current_node = to_expand.front();
            to_expand.pop_front();

            current_node->create_children(to_expand);

            n_nodes_to_create -= AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE;
            VLOG(11) << "Nodes to create: " << n_nodes_to_create;
            VLOG(11) << "Nodes in queue: " << to_expand.size();
        }

        VLOG(10) << "Done expanding hierarchy of n levels.";
    }

    void DeviceHierarchicalGaussianMixture::run_em() {
        auto ptr = std::shared_ptr<DevicePointCloud>(new DevicePointCloud(pcl));
        PointCloud vanilla_pcl(ptr);

        auto em = EmAlgorithm::from_pcl(vanilla_pcl);
        em.run(EM_CONVERGENCE_THRESHOLD);
    }

    void DeviceHierarchicalGaussianMixture::create_children(std::deque<DeviceHierarchicalGaussianMixture*>& to_expand) {
        VLOG(10) << "Creating children of hierarchical gaussian mixture...";

        SortByBestAssociationOperation op;
        std::vector<DevicePointCloud::PointIterator> boundaries = pcl.execute_pointcloud_operation(op);

        for(int i=0; i < boundaries.size() - 1; i++) {
            DevicePointCloud child_pcl;
            child_pcl.set_points(pcl.get_data(), boundaries[i], boundaries[i+1]);

            children.push_back(create_one_child(child_pcl, mixture.get_gaussian(i)));
            to_expand.push_back(&children.back());
        }

        VLOG(10) << "Done creating children";
    }

    DeviceHierarchicalGaussianMixture DeviceHierarchicalGaussianMixture::create_one_child(const DevicePointCloud& child_pcl,
                                                                                          const WeightedGaussian& parent_distribution) const {
        GaussianMixtureFactory gmm_factory;

        GaussianMixture child_mixture = gmm_factory.around_point(parent_distribution.get_mu(),
                                                                 parent_distribution.get_sigma(),
                                                                 AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE,
                                                                 UNIFORM_DISTRIBUTION_SIZE);

        std::cout << "ChildMixture: " << child_mixture;

        DeviceHierarchicalGaussianMixture child(child_pcl, child_mixture);
        child.run_em();

        return child;
    }
}
