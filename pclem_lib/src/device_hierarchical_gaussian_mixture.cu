#include "hip/hip_runtime.h"

#include <memory>
#include <glog/logging.h>
#include <thrust/partition.h>
#include <thrust/find.h>
#include <deque>

#include "em_algorithm.h"
#include "associated_point.cuh"
#include "device_hierarchical_gaussian_mixture.h"
#include "pointcloud.h"
#include "gaussian_mixture_factory.h"
#include "sort_by_best_association_operation.h"

namespace pclem {
    DeviceHierarchicalGaussianMixture::DeviceHierarchicalGaussianMixture(const DevicePointCloud& pcl, const GaussianMixture& mixture)
        : pcl(pcl), mixture(mixture), children() {}

    void DeviceHierarchicalGaussianMixture::expand_n_levels(int n_levels) {
        VLOG(10) << "Expanding hierarchy of n levels...";

        run_em();

        int n_nodes_to_create = std::pow(AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE, n_levels);
        std::deque<std::shared_ptr<DeviceHierarchicalGaussianMixture>> to_expand;

        this->create_children(to_expand);

        while(n_nodes_to_create > 0) {
            std::shared_ptr<DeviceHierarchicalGaussianMixture> current_node = to_expand.front();
            to_expand.pop_front();

            current_node->create_children(to_expand);

            n_nodes_to_create -= AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE;
            VLOG(11) << "Nodes to create: " << n_nodes_to_create;
            VLOG(11) << "Nodes in queue: " << to_expand.size();
        }

        VLOG(10) << "Done expanding hierarchy of n levels.";
    }

    void DeviceHierarchicalGaussianMixture::run_em() {
        VLOG(10) << "Running em of DeviceHierarchicalGaussianMixture...";

        auto ptr = std::shared_ptr<DevicePointCloud>(new DevicePointCloud(pcl));
        PointCloud vanilla_pcl(ptr);

        EmAlgorithm em(vanilla_pcl, mixture);
        em.run(EM_CONVERGENCE_THRESHOLD);

        mixture = em.get_mixture();

        VLOG(10) << "Done running em of DeviceHierarchicalGaussianMixture";
    }

    void DeviceHierarchicalGaussianMixture::create_children(std::deque<std::shared_ptr<DeviceHierarchicalGaussianMixture>>& to_expand) {
        VLOG(10) << "Creating children of hierarchical gaussian mixture...";

        SortByBestAssociationOperation op;
        auto boundaries = pcl.execute_pointcloud_operation(op);

        for(int i=0; i < boundaries.size() - 1; i++) {
            DevicePointCloud child_pcl;

            std::cout << "Boundaries found: " << boundaries[i] << " ";
            std::cout << boundaries[i+1] << std::endl;

            child_pcl.set_points(pcl.get_data(), pcl.begin() + boundaries[i], pcl.begin() + boundaries[i+1]);

            WeightedGaussian current_gaussian = mixture.get_gaussian(i);

            if(current_gaussian.get_weight() > MIN_WEIGHT_TO_PROCREATE) {
                GaussianMixtureFactory gmm_factory;
                GaussianMixture child_mixture = gmm_factory.around_point(current_gaussian.get_mu(),
                                                                         current_gaussian.get_sigma(),
                                                                         AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE,
                                                                         UNIFORM_DISTRIBUTION_SIZE);

                auto child = std::shared_ptr<DeviceHierarchicalGaussianMixture>(new DeviceHierarchicalGaussianMixture(child_pcl, child_mixture));
                child->run_em();

                children.push_back(child);
                to_expand.push_back(child);
            }
        }

        VLOG(10) << "Done creating children";
    }

    DeviceHierarchicalGaussianMixture DeviceHierarchicalGaussianMixture::create_one_child(const DevicePointCloud& child_pcl,
                                                                                          const WeightedGaussian& parent_distribution) const {
        VLOG(10) << "Creating child of hierarchical gaussian mixture...";

        GaussianMixtureFactory gmm_factory;

        GaussianMixture child_mixture = gmm_factory.around_point(parent_distribution.get_mu(),
                                                                 parent_distribution.get_sigma(),
                                                                 AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE,
                                                                 UNIFORM_DISTRIBUTION_SIZE);

        DeviceHierarchicalGaussianMixture child(child_pcl, child_mixture);
        child.run_em();

        VLOG(10) << "Don creating child of hierarchical gaussian mixture.";
        return child;
    }

    std::ostream& operator<<(std::ostream& os, const DeviceHierarchicalGaussianMixture& hierarchy) {
        hierarchy.print_with_padding(os, 0);
        return os;
    }

    void DeviceHierarchicalGaussianMixture::print_with_padding(std::ostream& os, int padding) const {
        for(int i = 0; i < padding; i++) {
            os << " ";
        }
        os << mixture.n_gaussians() << std::endl;

        for(std::shared_ptr<DeviceHierarchicalGaussianMixture> child: children) {
            child->print_with_padding(os, padding+1);
        }
    }
}
