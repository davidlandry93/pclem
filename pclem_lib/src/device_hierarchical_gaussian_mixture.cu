#include "hip/hip_runtime.h"

#include <memory>
#include <glog/logging.h>
#include <thrust/partition.h>
#include <thrust/find.h>
#include <deque>

#include "em_algorithm.h"
#include "associated_point.cuh"
#include "device_hierarchical_gaussian_mixture.h"
#include "pointcloud.h"
#include "gaussian_mixture_factory.h"
#include "sort_by_best_association_operation.h"

namespace pclem {
    DeviceHierarchicalGaussianMixture::DeviceHierarchicalGaussianMixture(const DevicePointCloud& pcl, const GaussianMixture& mixture)
        : pcl(pcl), mixture(mixture), children() {}

    void DeviceHierarchicalGaussianMixture::expand_n_levels(int n_levels) {
        VLOG(10) << "Expanding hierarchy of n levels...";

        run_em();

        int n_nodes_to_create = std::pow(AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE, n_levels);
        std::deque<DeviceHierarchicalGaussianMixture*> to_expand;

        to_expand.push_back(this);
        while(n_nodes_to_create > 0) {
            DeviceHierarchicalGaussianMixture* current_node = to_expand.front();
            to_expand.pop_front();

            current_node->create_children(to_expand);

            n_nodes_to_create -= AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE;
            VLOG(11) << "Nodes to create: " << n_nodes_to_create;
            VLOG(11) << "Nodes in queue: " << to_expand.size();
        }

        VLOG(10) << "Done expanding hierarchy of n levels.";
    }

    void DeviceHierarchicalGaussianMixture::run_em() {
        VLOG(10) << "Running em of DeviceHierarchicalGaussianMixture...";

        auto ptr = std::shared_ptr<DevicePointCloud>(new DevicePointCloud(pcl));
        PointCloud vanilla_pcl(ptr);

        EmAlgorithm em(vanilla_pcl, mixture);
        em.run(EM_CONVERGENCE_THRESHOLD);

        mixture = em.get_mixture();

        VLOG(10) << "Done running em of DeviceHierarchicalGaussianMixture";
    }

    void DeviceHierarchicalGaussianMixture::create_children(std::deque<DeviceHierarchicalGaussianMixture*>& to_expand) {
        VLOG(10) << "Creating children of hierarchical gaussian mixture...";

        SortByBestAssociationOperation op;
        auto boundaries = pcl.execute_pointcloud_operation(op);

        for(int i=0; i < boundaries.size() - 1; i++) {
            DevicePointCloud child_pcl;
            child_pcl.set_points(pcl.get_data(), pcl.begin() + boundaries[i], pcl.begin() + boundaries[i+1]);

            WeightedGaussian current_gaussian = mixture.get_gaussian(i);

            if(current_gaussian.get_weight() > MIN_WEIGHT_TO_PROCREATE) {
                children.push_back(create_one_child(child_pcl, current_gaussian));
                to_expand.push_back(&children.back());
            }
        }

        VLOG(10) << "Done creating children";
    }

    DeviceHierarchicalGaussianMixture DeviceHierarchicalGaussianMixture::create_one_child(const DevicePointCloud& child_pcl,
                                                                                          const WeightedGaussian& parent_distribution) const {
        VLOG(10) << "Creating child of hierarchical gaussian mixture...";

        GaussianMixtureFactory gmm_factory;

        GaussianMixture child_mixture = gmm_factory.around_point(parent_distribution.get_mu(),
                                                                 parent_distribution.get_sigma(),
                                                                 AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE,
                                                                 UNIFORM_DISTRIBUTION_SIZE);

        DeviceHierarchicalGaussianMixture child(child_pcl, child_mixture);
        child.run_em();

        VLOG(10) << "Don creating child of hierarchical gaussian mixture.";
        return child;
    }
}
