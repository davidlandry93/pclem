#include "hip/hip_runtime.h"

#include <glog/logging.h>

#include "device_pointcloud.h"
#include "covariance_matrix.h"
#include "em_algorithm.h"
#include "gaussian_mixture_factory.h"

namespace pclem {
    EmAlgorithm::EmAlgorithm(PointCloud& pcl,
                             GaussianMixture& mixture) :
        pcl(std::move(pcl)),
        mixture(std::move(mixture)) {}

    EmAlgorithm::EmAlgorithm(EmAlgorithm&& other) :
        pcl(std::move(other.pcl)), mixture(std::move(other.mixture)) {}

    EmAlgorithm EmAlgorithm::from_pcl(PointCloud& pcl) {
        GaussianMixtureFactory gm_factory;
        GaussianMixture mixture = gm_factory.from_pcl_corners(pcl);

        EmAlgorithm temp_em(pcl, mixture);

        return temp_em;
    }

    EmAlgorithm& EmAlgorithm::operator=(EmAlgorithm&& other) {
        std::swap(pcl, other.pcl);
        std::swap(mixture, other.mixture);
        return *this;
    }

    void EmAlgorithm::expectation() {
        VLOG(10) << "Computing expectation...";

        pcl.compute_associations(mixture);
        pcl.normalize_associations();

        VLOG(10) << "Done.";
    }

    void EmAlgorithm::maximization() {
        VLOG(10) << "Computing maximization...";
        mixture = pcl.create_mixture();
        VLOG(10) << "Done.";
    }

    void EmAlgorithm::run(double epsilon) {
        VLOG(10) << "Running expectation maximization...";

        double previous_likelihood = 0.0;
        double delta = std::numeric_limits<double>::infinity();

        while(delta > epsilon ||
              previous_likelihood == -1*std::numeric_limits<double>::infinity()) {
            expectation();
            maximization();

            double new_likelihood = log_likelihood();
            delta = std::abs(new_likelihood - previous_likelihood);
            LOG(INFO) << "Log likelihood: " << new_likelihood;
            LOG(INFO) << "Delta: " << delta;

            previous_likelihood = new_likelihood;
        }

        VLOG(10) << "Done running expectation maximization...";
    }

    GaussianMixture EmAlgorithm::get_mixture() const {
        return mixture;
    }

    std::ostream& operator<<(std::ostream& os, const EmAlgorithm& em) {
        os << "===EM===" << std::endl << em.mixture;
        return os;
    }

    double EmAlgorithm::log_likelihood() {
        return pcl.log_likelihood_of_mixture(mixture);
    }
}
