#include "hip/hip_runtime.h"

#include <glog/logging.h>

#include "device_pointcloud.h"
#include "em_algorithm.h"
#include "gaussian_mixture_factory.h"

namespace pclem {
    EmAlgorithm::EmAlgorithm(PointCloud& pcl,
                             GaussianMixture& mixture) :
        pcl(std::move(pcl)),
        mixture(std::move(mixture)) {}

    EmAlgorithm::EmAlgorithm(EmAlgorithm&& other) :
        pcl(std::move(other.pcl)), mixture(std::move(other.mixture)) {}

    EmAlgorithm EmAlgorithm::from_pcl(PointCloud& pcl) {
        GaussianMixtureFactory gm_factory;
        GaussianMixture mixture = gm_factory.from_pcl_corners(pcl, 1.0);

        EmAlgorithm temp_em(pcl, mixture);

        return temp_em;
    }

    EmAlgorithm& EmAlgorithm::operator=(EmAlgorithm&& other) {
        std::swap(pcl, other.pcl);
        std::swap(mixture, other.mixture);
        return *this;
    }

    void EmAlgorithm::expectation() {
        VLOG(10) << "Computing expectation...";

        pcl.compute_associations(mixture);
        pcl.normalize_associations();

        VLOG(10) << "Done.";
    }

    void EmAlgorithm::maximization() {
        VLOG(10) << "Computing maximization...";
        mixture = pcl.create_mixture();
        VLOG(10) << "Done.";
    }

    void EmAlgorithm::run(double epsilon) {
        VLOG(10) << "Running expectation maximization...";

        std::cout << "Running em..." << std::endl;
        std::cout << "PCL has " << pcl.get_n_points() << " points" << std::endl;
        std::cout << "Initial mixture: " << std::endl << mixture << std::endl;

        double previous_likelihood = 0.0;
        double delta = std::numeric_limits<double>::infinity();
        int n_iterations = 0;

        while((delta > epsilon ||
               previous_likelihood == -1*std::numeric_limits<double>::infinity() ||
               n_iterations < MIN_N_ITERATIONS) &&
              n_iterations < MAX_N_ITERATIONS) {
            expectation();
            maximization();

            double new_likelihood = log_likelihood();
            delta = std::abs(new_likelihood - previous_likelihood);
            VLOG(3) << "Log likelihood: " << new_likelihood;
            VLOG(3) << "Delta: " << delta;

            previous_likelihood = new_likelihood;

            n_iterations++;
        }

        std::cout << "Result: " << std::endl << mixture;

        VLOG(10) << "Done running expectation maximization...";
    }

    GaussianMixture EmAlgorithm::get_mixture() const {
        return mixture;
    }

    std::ostream& operator<<(std::ostream& os, const EmAlgorithm& em) {
        os << "===EM===" << std::endl << em.mixture;
        return os;
    }

    double EmAlgorithm::log_likelihood() {
        return pcl.log_likelihood_of_mixture(mixture);
    }
}
