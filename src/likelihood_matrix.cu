#include "hip/hip_runtime.h"

#include <thrust/extrema.h>
#include <armadillo>
#include "likelihood_matrix.h"
#include "strided_range.h"

namespace pclem {
    LikelihoodMatrix::LikelihoodMatrix() : likelihoods() {

    }

    LikelihoodMatrix::LikelihoodMatrix(int n_points,
                                       int n_distributions,
                                       std::vector<double>& likelihoods)
        : n_points(n_points), n_distributions(n_distributions), likelihoods(likelihoods) {}

    LikelihoodMatrix::LikelihoodMatrix(int n_points,
                                       int n_distributions,
                                       thrust::device_vector<double>& likelihoods)
        : n_points(n_points), n_distributions(n_distributions), likelihoods(likelihoods) {}

    LikelihoodMatrix::LikelihoodMatrix(LikelihoodMatrix&& other) : likelihoods() {
        std::swap(likelihoods, other.likelihoods);
    }

    LikelihoodMatrix LikelihoodMatrix::build(const PointCloud& pcl, const GaussianMixture& mixture) {
        int n_points = pcl.get_n_points();
        int n_gaussians = mixture.n_gaussians();
        thrust::device_vector<double> likelihoods(n_points*n_gaussians);

        for(int i = 0; i < n_gaussians; i++) {
            likelihoods_of_distribution(pcl, mixture.get_gaussian(i), likelihoods.begin() + i*n_points);
        }

        normalize_likelihoods(n_points, likelihoods);

        return LikelihoodMatrix(n_points, n_gaussians, likelihoods);
    }

    LikelihoodMatrix& LikelihoodMatrix::operator=(LikelihoodMatrix&& other) {
        std::swap(likelihoods, other.likelihoods);
        return *this;
    }

    struct gaussian_op : public thrust::unary_function<Point,double> {
    public:
        gaussian_op(Point _mu, double _det, std::array<double,9> _inv_of_covariance) :
            mu(_mu),
            base(1 / sqrt(pow(2*M_PI, 3) * _det)),
            inv() {
            for(int i = 0; i < 9; i++) {
                inv[i] = _inv_of_covariance[i];
            }

            std::cout << _mu;
            std::cout << _det;
        }


        // https://en.wikipedia.org/wiki/Multivariate_normal_distribution#Density_function
        __host__ __device__
        double operator()(Point x) {
            Point x_minus_mu = x - mu;
            double temp_product[3] = {x_minus_mu.z*inv[6] + x_minus_mu.y*inv[3] + x_minus_mu.x*inv[0],
                                      x_minus_mu.z*inv[7] + x_minus_mu.y*inv[4] + x_minus_mu.x*inv[1],
                                      x_minus_mu.z*inv[8] + x_minus_mu.y*inv[5] + x_minus_mu.x*inv[2]};

            double scale_product =
                x_minus_mu.x*temp_product[0] +
                x_minus_mu.y*temp_product[1] +
                x_minus_mu.z*temp_product[2];

            return base * exp(-0.5 * scale_product);
        }

        __const__ Point mu;
        __const__ double base;
        double inv[9];
    };

    void LikelihoodMatrix::likelihoods_of_distribution(const PointCloud& pcl,
                                                       const WeightedGaussian& distribution,
                                                       thrust::device_vector<double>::iterator result) {
        std::array<double,9> cov_mat = distribution.get_sigma().as_array();

        arma::mat33 arma_cov_mat(cov_mat.data());
        double det_of_covariance = arma::det(arma_cov_mat);

        arma::mat33 arma_inv_of_covariance = arma::inv_sympd(arma_cov_mat);

        std::array<double,9> inv_of_covariance;
        for(auto i = 0; i < 3; i++) {
            for (auto j=0; j < 3; j++) {
                inv_of_covariance[i*3+j] = arma_inv_of_covariance(i,j);
            }
        }

        gaussian_op op(distribution.get_mu(), det_of_covariance, inv_of_covariance);

        thrust::transform(pcl.begin(), pcl.end(), result, op);
    }

    void LikelihoodMatrix::normalize_likelihoods(int n_points,
                                                 thrust::device_vector<double>& likelihoods) {
        typedef StridedRange<thrust::device_vector<double>::iterator> strided_iterator;

        for(int i = 0; i < n_points; i++) {
            std::cout << "Before iterator" << std::endl;
            strided_iterator iterator(likelihoods.begin() + i, likelihoods.end(), n_points);

            for(auto it = iterator.begin(); it != iterator.end(); it++) {
                std::cout << *it;
            }
            std::cout << std::endl;
        }
    }
}
