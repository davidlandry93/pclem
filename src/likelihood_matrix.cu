#include "hip/hip_runtime.h"

#include <thrust/iterator/constant_iterator.h>
#include <thrust/inner_product.h>
#include <thrust/extrema.h>
#include <armadillo>
#include "likelihood_matrix.h"
#include "strided_range.h"

namespace pclem {
    LikelihoodMatrix::LikelihoodMatrix() : likelihoods() {

    }

    LikelihoodMatrix::LikelihoodMatrix(int n_points,
                                       int n_distributions,
                                       std::vector<double>& likelihoods)
        : n_points(n_points), n_distributions(n_distributions), likelihoods(likelihoods) {}

    LikelihoodMatrix::LikelihoodMatrix(int n_points,
                                       int n_distributions,
                                       thrust::device_vector<double>& likelihoods)
        : n_points(n_points), n_distributions(n_distributions), likelihoods(likelihoods) {}

    LikelihoodMatrix::LikelihoodMatrix(LikelihoodMatrix&& other) : likelihoods() {
        std::swap(likelihoods, other.likelihoods);
    }

    LikelihoodMatrix LikelihoodMatrix::build(const PointCloud& pcl, const GaussianMixture& mixture) {
        int n_points = pcl.get_n_points();
        int n_gaussians = mixture.n_gaussians();
        thrust::device_vector<double> likelihoods(n_points*n_gaussians);

        for(int i = 0; i < n_gaussians; i++) {
            likelihoods_of_distribution(pcl, mixture.get_gaussian(i), likelihoods.begin() + i*n_points);
        }

        normalize_likelihoods(n_points, likelihoods);

        return LikelihoodMatrix(n_points, n_gaussians, likelihoods);
    }

    LikelihoodMatrix& LikelihoodMatrix::operator=(LikelihoodMatrix&& other) {
        std::swap(likelihoods, other.likelihoods);
        return *this;
    }

    struct gaussian_op : public thrust::unary_function<Point,double> {
    public:
        gaussian_op(Point _mu, double _det, std::array<double,9> _inv_of_covariance) :
            mu(_mu),
            base(1 / sqrt(pow(2*M_PI, 3) * _det)),
            inv() {
            for(int i = 0; i < 9; i++) {
                inv[i] = _inv_of_covariance[i];
            }
        }


        // https://en.wikipedia.org/wiki/Multivariate_normal_distribution#Density_function
        __host__ __device__
        double operator()(Point x) {
            Point x_minus_mu = x - mu;
            double temp_product[3] = {x_minus_mu.z*inv[6] + x_minus_mu.y*inv[3] + x_minus_mu.x*inv[0],
                                      x_minus_mu.z*inv[7] + x_minus_mu.y*inv[4] + x_minus_mu.x*inv[1],
                                      x_minus_mu.z*inv[8] + x_minus_mu.y*inv[5] + x_minus_mu.x*inv[2]};

            double scale_product =
                x_minus_mu.x*temp_product[0] +
                x_minus_mu.y*temp_product[1] +
                x_minus_mu.z*temp_product[2];

            return base * exp(-0.5 * scale_product);
        }

        __const__ Point mu;
        __const__ double base;
        double inv[9];
    };

    void LikelihoodMatrix::likelihoods_of_distribution(const PointCloud& pcl,
                                                       const WeightedGaussian& distribution,
                                                       thrust::device_vector<double>::iterator result) {
        std::array<double,9> cov_mat = distribution.get_sigma().as_array();

        arma::mat33 arma_cov_mat(cov_mat.data());
        double det_of_covariance = arma::det(arma_cov_mat);

        arma::mat33 arma_inv_of_covariance = arma::inv_sympd(arma_cov_mat);

        std::array<double,9> inv_of_covariance;
        for(auto i = 0; i < 3; i++) {
            for (auto j=0; j < 3; j++) {
                inv_of_covariance[i*3+j] = arma_inv_of_covariance(i,j);
            }
        }

        gaussian_op op(distribution.get_mu(), det_of_covariance, inv_of_covariance);

        thrust::transform(pcl.begin(), pcl.end(), result, op);
    }

    void LikelihoodMatrix::normalize_likelihoods(int n_points,
                                                 thrust::device_vector<double>& likelihoods) {
        typedef StridedRange<thrust::device_vector<double>::iterator> strided_iterator;

        for(int i = 0; i < n_points; i++) {
            strided_iterator iterator(likelihoods.begin() + i, likelihoods.end(), n_points);

            double sum_of_likelihoods = thrust::reduce(iterator.begin(), iterator.end(), 0.0, thrust::plus<double>());

            thrust::transform(iterator.begin(), iterator.end(),
                              thrust::make_constant_iterator(sum_of_likelihoods),
                              iterator.begin(), thrust::divides<double>());
        }
    }

    struct point_by_scalar_op : public thrust::binary_function<Point,double,Point> {
    public:
        __host__ __device__
        Point operator()(Point x, double pi) {
            return Point(x.x * pi, x.y * pi, x.z * pi);
        }
    };

    struct sum_of_pts_op : public thrust::binary_function<Point,Point,Point> {
    public:
        __host__ __device__
        Point operator()(Point lhs, Point rhs) {
            return Point(lhs.x + rhs.x,
                         lhs.y + rhs.y,
                         lhs.z + rhs.z);
        }
    };

    GaussianMixture LikelihoodMatrix::gaussian_mixture_of_pcl(const PointCloud& pcl) const {
        std::vector<WeightedGaussian> gaussians;
        thrust::device_vector<Point> mus(n_distributions);

        for(int i = 0; i < n_distributions; i++) {
            Point new_mu = thrust::inner_product(pcl.begin(), pcl.end(),
                                                 likelihoods.begin() + i*n_points,
                                                 Point(0.0, 0.0, 0.0),
                                                 sum_of_pts_op(),
                                                 point_by_scalar_op());

            double sum_of_gammas = thrust::reduce(likelihoods.begin() + i*n_points,
                                                  likelihoods.begin() + (i+1)*n_points,
                                                  0.0, thrust::plus<double>());

            new_mu = Point(new_mu.x / sum_of_gammas,
                           new_mu.y / sum_of_gammas,
                           new_mu.z / sum_of_gammas);

            std::cout << "New mu: " << new_mu <<
                "Sum of gammas: " << sum_of_gammas << std::endl;
        }

        return GaussianMixture(gaussians);
    }
}
