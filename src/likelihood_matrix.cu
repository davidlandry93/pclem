#include "hip/hip_runtime.h"

#include <math.h>
#include <array>
#include <glog/logging.h>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/inner_product.h>
#include <thrust/extrema.h>
#include <armadillo>
#include "likelihood_matrix.h"
#include "strided_range.h"

namespace pclem {
    LikelihoodMatrix::LikelihoodMatrix() : likelihoods() {

    }

    LikelihoodMatrix::LikelihoodMatrix(int n_points,
                                       int n_distributions,
                                       std::vector<double>& likelihoods)
        : n_points(n_points), n_distributions(n_distributions), likelihoods(likelihoods) {}

    LikelihoodMatrix::LikelihoodMatrix(int n_points,
                                       int n_distributions,
                                       thrust::device_vector<double>& likelihoods)
        : n_points(n_points), n_distributions(n_distributions), likelihoods(likelihoods) {}

    LikelihoodMatrix::LikelihoodMatrix(LikelihoodMatrix&& other) : likelihoods() {
        std::swap(likelihoods, other.likelihoods);
    }

    LikelihoodMatrix LikelihoodMatrix::build(const PointCloud& pcl, const GaussianMixture& mixture) {
        int n_points = pcl.get_n_points();
        int n_gaussians = mixture.n_gaussians();
        thrust::device_vector<double> likelihoods(n_points*n_gaussians);

        for(int i = 0; i < n_gaussians; i++) {
            likelihoods_of_distribution(pcl, mixture.get_gaussian(i), likelihoods.begin() + i*n_points);
        }

        normalize_likelihoods(n_points, likelihoods);

        return LikelihoodMatrix(n_points, n_gaussians, likelihoods);
    }

    LikelihoodMatrix& LikelihoodMatrix::operator=(LikelihoodMatrix&& other) {
        std::swap(likelihoods, other.likelihoods);
        return *this;
    }

    struct gaussian_op : public thrust::unary_function<Point,double> {
    public:
        gaussian_op(Point _mu, double _weight, double _det, std::array<double,9> _inv_of_covariance) :
            mu(_mu),
            base(_weight / sqrt(pow(2*M_PI, 3) * _det)),
            inv() {
            for(int i = 0; i < 9; i++) {
                inv[i] = _inv_of_covariance[i];
            }
        }


        // https://en.wikipedia.org/wiki/Multivariate_normal_distribution#Density_function
        __host__ __device__
        double operator()(Point x) {
            Point x_minus_mu = x - mu;
            double temp_product[3] = {x_minus_mu.z*inv[6] + x_minus_mu.y*inv[3] + x_minus_mu.x*inv[0],
                                      x_minus_mu.z*inv[7] + x_minus_mu.y*inv[4] + x_minus_mu.x*inv[1],
                                      x_minus_mu.z*inv[8] + x_minus_mu.y*inv[5] + x_minus_mu.x*inv[2]};

            double scale_product =
                x_minus_mu.x*temp_product[0] +
                x_minus_mu.y*temp_product[1] +
                x_minus_mu.z*temp_product[2];

            return base * exp(-0.5 * scale_product);
        }

        __const__ Point mu;
        __const__ double base;
        double inv[9];
    };

    void LikelihoodMatrix::likelihoods_of_distribution(const PointCloud& pcl,
                                                       const WeightedGaussian& distribution,
                                                       thrust::device_vector<double>::iterator result) {
        VLOG(10) << "Computing likelihoods...";
        std::array<double,9> inv_cov_mat = distribution.get_sigma().inverse();

        gaussian_op op(distribution.get_mu(), distribution.get_weight(), distribution.get_sigma().det(), inv_cov_mat);

        thrust::transform(pcl.begin(), pcl.end(), result, op);
        VLOG(10) << "Done computing likelihoods.";
    }

    void LikelihoodMatrix::normalize_likelihoods(int n_points,
                                                 thrust::device_vector<double>& likelihoods) {
        VLOG(10) << "Normalizing likelihoods...";

        typedef StridedRange<thrust::device_vector<double>::iterator> strided_iterator;

        for(int i = 0; i < n_points; i++) {
            strided_iterator iterator(likelihoods.begin() + i, likelihoods.end(), n_points);

            double sum_of_likelihoods = thrust::reduce(iterator.begin(), iterator.end(),
                                                       0.0, thrust::plus<double>());

            thrust::transform(iterator.begin(), iterator.end(),
                              thrust::make_constant_iterator(sum_of_likelihoods),
                              iterator.begin(), thrust::divides<double>());
        }

        VLOG(10) << "Done.";
    }



    GaussianMixture LikelihoodMatrix::gaussian_mixture_of_pcl(const PointCloud& pcl) const {
        std::vector<WeightedGaussian> gaussians;
        thrust::device_vector<Point> mus(n_distributions);

        for(int i = 0; i < n_distributions; i++) {
            double sum_of_gammas = thrust::reduce(likelihoods.begin() + i*n_points,
                                                  likelihoods.begin() + (i+1)*n_points,
                                                  0.0, thrust::plus<double>());

            Point new_mu = compute_mu(pcl, likelihoods.begin() + i*n_points, sum_of_gammas);
            CovarianceMatrix new_sigma = compute_sigma(pcl,
                                                       likelihoods.begin() + i*n_points,
                                                       sum_of_gammas,
                                                       new_mu);

            double new_weight = sum_of_gammas / n_points;

            WeightedGaussian to_add(new_mu, new_sigma, new_weight);

            std::cout << "Added gaussian " << i << ": " << std::endl
                      << to_add;

            gaussians.push_back(to_add);
        }

        return GaussianMixture(gaussians);
    }

    struct point_by_scalar_op : public thrust::binary_function<Point,double,Point> {
    public:
        __host__ __device__
        Point operator()(Point x, double pi) {
            return Point(x.x * pi, x.y * pi, x.z * pi);
        }
    };

    struct sum_of_pts_op : public thrust::binary_function<Point,Point,Point> {
    public:
        __host__ __device__
        Point operator()(Point lhs, Point rhs) {
            return Point(lhs.x + rhs.x,
                         lhs.y + rhs.y,
                         lhs.z + rhs.z);
        }
    };

    Point LikelihoodMatrix::compute_mu(const PointCloud& pcl,
                                       thrust::device_vector<double>::const_iterator likelihoods,
                                       double sum_of_gammas) const {
        Point new_mu = thrust::inner_product(pcl.begin(), pcl.end(),
                                             likelihoods,
                                             Point(0.0, 0.0, 0.0),
                                             sum_of_pts_op(),
                                             point_by_scalar_op());

        new_mu = Point(new_mu.x / sum_of_gammas,
                       new_mu.y / sum_of_gammas,
                       new_mu.z / sum_of_gammas);

        return new_mu;
    }

    struct sum_of_cov_matrix_op : public thrust::binary_function<RawCovarianceMatrix, RawCovarianceMatrix, RawCovarianceMatrix> {
    public:
        __host__ __device__
        RawCovarianceMatrix operator()(RawCovarianceMatrix lhs, RawCovarianceMatrix rhs) {
            return lhs + rhs;
        }
    };

    struct point_to_cov_op : public thrust::binary_function<Point,double,RawCovarianceMatrix> {
    public:
        __host__ __device__
        RawCovarianceMatrix operator()(Point& p, double gamma) {
            RawCovarianceMatrix r;

            r.v00 = p.x * p.x * gamma;
            r.v11 = p.y * p.y * gamma;
            r.v22 = p.z * p.z * gamma;

            r.v10 = r.v01 = p.x * p.y * gamma;
            r.v20 = r.v02 = p.x * p.z * gamma;
            r.v21 = r.v12 = p.y * p.z * gamma;

            return r;
        }
    };

    CovarianceMatrix LikelihoodMatrix::compute_sigma(const PointCloud& pcl,
                                                        thrust::device_vector<double>::const_iterator likelihoods,
                                                        double sum_of_gammas,
                                                        const Point& new_mu) const {
        RawCovarianceMatrix init = RawCovarianceMatrix::zeros();
        RawCovarianceMatrix new_sigma = thrust::inner_product(pcl.begin(), pcl.end(),
                                                              likelihoods,
                                                              init,
                                                              sum_of_cov_matrix_op(),
                                                              point_to_cov_op());

        RawCovarianceMatrix base_sigma;

        base_sigma.v00 = new_mu.x * new_mu.x;
        base_sigma.v11 = new_mu.y * new_mu.y;
        base_sigma.v22 = new_mu.z * new_mu.z;

        base_sigma.v01 = base_sigma.v10 = new_mu.x * new_mu.y;
        base_sigma.v02 = base_sigma.v20 = new_mu.x * new_mu.z;
        base_sigma.v21 = base_sigma.v12 = new_mu.z * new_mu.y;

        new_sigma = new_sigma / sum_of_gammas - base_sigma;

        return CovarianceMatrix(new_sigma);
    }


    double LikelihoodMatrix::log_likelihood(const PointCloud& pcl, const GaussianMixture& mixture) const {
        VLOG(10) << "Computing log likelihood...";

        double log_likelihood = 0.0;
        for(auto i = 0; i < n_distributions; i++) {
            double to_add = log_likelihood_of_distribution(pcl, mixture.get_gaussian(i),
                                                           likelihoods.begin() + i*n_points);

            std::cout << to_add << " ";

            log_likelihood += to_add;
        }

        VLOG(10) << "Done computing log likelihood.";
        return log_likelihood;
    }

    struct log_likelihood_op : public thrust::binary_function<Point, double, double> {
    public:
        log_likelihood_op(const WeightedGaussian& distribution) :
            log_pi_ij(log(distribution.get_weight())),
            base(1.0 / sqrt(pow(2*M_PI, 3) * distribution.get_sigma().det())),
            mu(distribution.get_mu()),
            inv() {
            std::array<double,9> inv_of_sigma = distribution.get_sigma().inverse();

            for(int i =0; i < 9; i++) {
                inv[i] = inv_of_sigma[i];
            }
        }

        __host__ __device__
        double operator()(Point p, double gamma) {
            return gamma * (log_pi_ij + log(likelihood_of_point(p)));
        }

    private:
        __host__ __device__
        double likelihood_of_point(Point p) {
            Point x_minus_mu = p - mu;

            double temp_product[3] = {x_minus_mu.z*inv[6] + x_minus_mu.y*inv[3] + x_minus_mu.x*inv[0],
                                      x_minus_mu.z*inv[7] + x_minus_mu.y*inv[4] + x_minus_mu.x*inv[1],
                                      x_minus_mu.z*inv[8] + x_minus_mu.y*inv[5] + x_minus_mu.x*inv[2]};

            double scale_product = x_minus_mu.x * temp_product[0] +
                x_minus_mu.y * temp_product[1] +
                x_minus_mu.z * temp_product[2];

            return base * exp(-0.5 * scale_product);
        }


        const double log_pi_ij;
        const double base;
        const Point mu;
        double inv[9];
    };


    double LikelihoodMatrix::log_likelihood_of_distribution(const PointCloud& pcl, const WeightedGaussian& distribution,
                                          thrust::device_vector<double>::const_iterator likelihoods) const {
        return thrust::inner_product(pcl.begin(), pcl.end(),
                                     likelihoods, 0.0,
                                     thrust::plus<double>(),
                                     log_likelihood_op(distribution));
    }
}
