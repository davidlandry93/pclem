#include "hip/hip_runtime.h"

#include <iostream>
#include <limits>
#include <cmath>

#include <vtkSmartPointer.h>
#include <vtkGenericDataObjectReader.h>
#include <vtkPolyData.h>
#include <glog/logging.h>

#include "pointcloud.h"
#include "em_algorithm.h"

using namespace pclem;

int main(int argc, char** argv) {
    google::InitGoogleLogging(argv[0]);

    LOG(INFO) << "Hello world";

    VLOG(10) << "Reading point cloud...";
    vtkSmartPointer<vtkGenericDataObjectReader> reader =
        vtkSmartPointer<vtkGenericDataObjectReader>::New();
    reader->SetFileName("res/foret.vtk");
    reader->Update();
    VLOG(10) << "Done.";

    vtkPolyData* output = reader->GetPolyDataOutput();

    PointCloud pcl = PointCloud::from_vtk(output);

    auto em = EmAlgorithm::from_pcl(pcl);

    std::cout << em;
    double previous_likelihood = 0.0;
    double delta = std::numeric_limits<double>::infinity();
    while (delta > 0.0001){
        em.expectation();
        em.maximization();

        double new_likelihood = em.log_likelihood();
        delta = std::abs(new_likelihood - previous_likelihood);
        std::cout << ". LOG LIKELIHOOD: " << new_likelihood << " Delta: " << delta << std::endl;

        previous_likelihood = new_likelihood;
    }
    std::cout << em;

    return 0;
}
