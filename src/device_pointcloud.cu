#include "hip/hip_runtime.h"

#include <math.h>
#include <cstdio>
#include <chrono>
#include <limits>
#include <glog/logging.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#include <armadillo>

#include "strided_range.h"
#include "device_pointcloud.h"
#include "raw_covariance_matrix.cuh"
#include "gaussian_mixture_factory.h"
#include "device_hierarchical_gaussian_mixture.h"

namespace pclem {
    DevicePointCloud::DevicePointCloud() :
        ptr_to_points(new thrust::device_vector<AssociatedPoint>()),
        pts_begin(ptr_to_points->begin()),
        pts_end(ptr_to_points->end()),
        boundingBox() {}

    DevicePointCloud::DevicePointCloud(const DevicePointCloud& other) :
        ptr_to_points(other.ptr_to_points),
        pts_begin(other.pts_begin),
        pts_end(other.pts_end),
        boundingBox(other.boundingBox){}

    BoundingBox DevicePointCloud::getBoundingBox() const {
        return BoundingBox(boundingBox);
    }

    struct min_op: public thrust::binary_function<DevicePoint,DevicePoint,DevicePoint> {
        __device__
        DevicePoint operator()(DevicePoint lhs, DevicePoint rhs) {
            return DevicePoint(thrust::min(lhs.x, rhs.x),
                               thrust::min(lhs.y, rhs.y),
                               thrust::min(lhs.z, rhs.z));
        }
    };

    struct max_op: public thrust::binary_function<DevicePoint,DevicePoint,DevicePoint> {
        __device__
        DevicePoint operator()(DevicePoint lhs, DevicePoint rhs) {
            return DevicePoint(thrust::max(lhs.x, rhs.x),
                               thrust::max(lhs.y, rhs.y),
                               thrust::max(lhs.z, rhs.z));
        }
    };

    void DevicePointCloud::updateBoundingBox(){
        min_op min_function;
        max_op max_function;

        DevicePoint min = thrust::reduce(pts_begin, pts_end, DevicePoint(0.0,0.0,0.0), min_function);
        DevicePoint max = thrust::reduce(pts_begin, pts_end, DevicePoint(0.0,0.0,0.0), max_function);

        Point host_min = min.to_host();
        Point host_max = max.to_host();

        boundingBox.setMin(host_min);
        boundingBox.setMax(host_max);
    }

    int DevicePointCloud::get_n_points() const {
        return pts_end - pts_begin;
    }

    void DevicePointCloud::set_points(const std::shared_ptr<thrust::device_vector<AssociatedPoint>>& points) {
        VLOG(10) << "Setting new data source...";

        ptr_to_points = points;
        pts_begin = points->begin();
        pts_end = points->end();

        updateBoundingBox();

        VLOG(10) << "Done setting data source.";
    }

    void DevicePointCloud::set_points(const std::shared_ptr<thrust::device_vector<AssociatedPoint>>& points,
                                      const PointIterator& begin,
                                      const PointIterator& end) {
        ptr_to_points = points;
        pts_begin = begin;
        pts_end = end;
    }

    void DevicePointCloud::compute_associations(const GaussianMixture& mixture) {
        VLOG(10) << "Computing point/distribution associations...";

        for(int i = 0; i < mixture.n_gaussians(); i++) {
            compute_associations_of_distribution(i, mixture.get_gaussian(i));
        }

        VLOG(10) << "Done computing point/distribution associations.";
    }

    struct gaussian_op : public thrust::unary_function<AssociatedPoint,AssociatedPoint> {
    public:
        gaussian_op(int index_of_distribution, const WeightedGaussian& distribution) :
            index_of_distribution(index_of_distribution),
            mu(distribution.get_mu()),
            base(distribution.get_weight() / sqrt(pow(2*M_PI,3) * distribution.get_sigma().det())),
            inv_of_covariance {0.0} {
            std::array<double,9> computed_inv = distribution.get_sigma().inverse();

            for (int i = 0; i < 9; i++) {
                inv_of_covariance[i] = computed_inv[i];
            }
        }

        __host__ __device__
        AssociatedPoint operator()(AssociatedPoint p) {
            p.likelihoods[index_of_distribution] = likelihood_of_point(p);
            return p;
        }

    private:
        __const__ int index_of_distribution;
        __const__ DevicePoint mu;
        __const__ double base;
        double inv_of_covariance[9];

        __host__ __device__
        double likelihood_of_point(DevicePoint p) {
            DevicePoint x_minus_mu = p - mu;

            double temp_product[3] = {x_minus_mu.z*inv_of_covariance[6] + x_minus_mu.y*inv_of_covariance[3] + x_minus_mu.x*inv_of_covariance[0],
                                      x_minus_mu.z*inv_of_covariance[7] + x_minus_mu.y*inv_of_covariance[4] + x_minus_mu.x*inv_of_covariance[1],
                                      x_minus_mu.z*inv_of_covariance[8] + x_minus_mu.y*inv_of_covariance[5] + x_minus_mu.x*inv_of_covariance[2]};

            double scale_product = x_minus_mu.x * temp_product[0] +
                x_minus_mu.y * temp_product[1] +
                x_minus_mu.z * temp_product[2];

            return base * exp(-0.5 * scale_product);
        }
    };

    void DevicePointCloud::compute_associations_of_distribution(int index_of_distribution, const WeightedGaussian& distribution) {
        gaussian_op op(index_of_distribution, distribution);

        thrust::transform(pts_begin, pts_end, pts_begin, op);
    }

    struct normalization_op : public thrust::unary_function<AssociatedPoint, AssociatedPoint> {
        __host__ __device__
        AssociatedPoint operator()(AssociatedPoint p) {
            double sum_of_associations = 0.0;
            for(int i = 0; i < p.N_DISTRIBUTIONS_PER_MIXTURE; i++) {
                sum_of_associations += p.likelihoods[i];
            }

            for(int i = 0; i < p.N_DISTRIBUTIONS_PER_MIXTURE; i++) {
                p.likelihoods[i] = p.likelihoods[i] / sum_of_associations;
            }

            return p;
        }
    };

    void DevicePointCloud::normalize_associations() {
        VLOG(10) << "Normalizing associations for every point...";

        thrust::transform(pts_begin, pts_end, pts_begin, normalization_op());

        VLOG(10) << "Done normalizing associations";
    }

    // This functor stores the sums of gammas in an AssociatedPoint.
    struct sums_of_gammas_op : public thrust::binary_function<AssociatedPoint, AssociatedPoint, AssociatedPoint> {

        __host__ __device__
        AssociatedPoint operator()(AssociatedPoint lhs, AssociatedPoint rhs) {
            AssociatedPoint sum;
            for(int i=0; i < AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE; i++) {
                sum.likelihoods[i] = lhs.likelihoods[i] + rhs.likelihoods[i];
            }
            return sum;
        }
    };

    GaussianMixture DevicePointCloud::create_mixture() const {
        // We store the sum of gammas of every distribution in an empty, meaningless AssociatedPoint.
        AssociatedPoint sums;
        sums = thrust::reduce(pts_begin, pts_end, AssociatedPoint(), sums_of_gammas_op());

        std::vector<WeightedGaussian> gaussians;

        for(int i=0; i < AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE; i++) {
            gaussians.push_back(create_distribution_of_mixture(i, sums.likelihoods[i]));
        }

        return GaussianMixture(gaussians);
    }

    struct weight_point_op : public thrust::unary_function<AssociatedPoint,DevicePoint> {
        int index_of_distribution;

        weight_point_op(int index_of_distribution) : index_of_distribution(index_of_distribution) {}

        __host__ __device__
        DevicePoint operator()(AssociatedPoint p) {
            return DevicePoint(p.x * p.likelihoods[index_of_distribution],
                               p.y * p.likelihoods[index_of_distribution],
                               p.z * p.likelihoods[index_of_distribution]);
        }
    };

    struct sum_of_points_op : public thrust::binary_function<DevicePoint,DevicePoint,DevicePoint> {
        __host__ __device__
        DevicePoint operator()(DevicePoint lhs, DevicePoint rhs) {
            return lhs + rhs;
        }
    };

    WeightedGaussian DevicePointCloud::create_distribution_of_mixture(int index_of_distribution, double sum_of_gammas) const {
        VLOG(10) << "Creating distribution " << index_of_distribution << " of mixture...";

        DevicePoint new_mu = thrust::transform_reduce(pts_begin, pts_end,
                                                      weight_point_op(index_of_distribution),
                                                      DevicePoint(0.0, 0.0, 0.0),
                                                      sum_of_points_op());

        new_mu = DevicePoint(new_mu.x / sum_of_gammas,
                             new_mu.y / sum_of_gammas,
                             new_mu.z / sum_of_gammas);

        VLOG(1) << "New mu: " << new_mu;

        CovarianceMatrix new_sigma = compute_sigma(index_of_distribution, new_mu.to_host(), sum_of_gammas);

        double new_weight = sum_of_gammas / get_n_points();

        VLOG(10) << "Done creating distribution " << index_of_distribution << " of mixture.";

        Point pub_new_mu = new_mu.to_host();
        WeightedGaussian to_return = WeightedGaussian(pub_new_mu, new_sigma, new_weight);

        return to_return;
    }

    struct point_to_cov_op : public thrust::unary_function<AssociatedPoint, RawCovarianceMatrix> {
        int index_of_distribution;

        point_to_cov_op(int index_of_distribution) : index_of_distribution(index_of_distribution) {}

        __host__ __device__
        RawCovarianceMatrix operator()(AssociatedPoint p) {
            RawCovarianceMatrix r;
            double gamma = p.likelihoods[index_of_distribution];

            r.v00 = p.x * p.x * gamma;
            r.v11 = p.y * p.y * gamma;
            r.v22 = p.z * p.z * gamma;

            r.v10 = r.v01 = p.x * p.y * gamma;
            r.v20 = r.v02 = p.x * p.z * gamma;
            r.v21 = r.v12 = p.y * p.z * gamma;

            return r;
        }
    };

    struct sum_of_cov_op : public thrust::binary_function<RawCovarianceMatrix, RawCovarianceMatrix, RawCovarianceMatrix> {
        __host__ __device__
        RawCovarianceMatrix operator()(RawCovarianceMatrix lhs, RawCovarianceMatrix rhs) {
            return lhs + rhs;
        }
    };

    CovarianceMatrix DevicePointCloud::compute_sigma(int index_of_distribution, const Point& mu, double sum_of_gammas) const {
        RawCovarianceMatrix init = RawCovarianceMatrix::zeros();

        RawCovarianceMatrix sigma = thrust::transform_reduce(pts_begin, pts_end,
                                                             point_to_cov_op(index_of_distribution),
                                                             init, sum_of_cov_op());

        RawCovarianceMatrix base_sigma;

        base_sigma.v00 = mu.x * mu.x;
        base_sigma.v11 = mu.y * mu.y;
        base_sigma.v22 = mu.z * mu.z;

        base_sigma.v01 = base_sigma.v10 = mu.x * mu.y;
        base_sigma.v02 = base_sigma.v20 = mu.x * mu.z;
        base_sigma.v21 = base_sigma.v12 = mu.z * mu.y;

        sigma = (sigma / sum_of_gammas) - base_sigma;
        return sigma.to_host();
    }

    double DevicePointCloud::log_likelihood_of_mixture(const GaussianMixture& mixture) const {
        double log_likelihood = 0.0;

        // std::cout << "Log likelihood of distributions: ";
        for(int i=0; i < AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE; i++) {
            double likelihood_of_distribution = log_likelihood_of_distribution(i, mixture.get_gaussian(i));

            // std::cout << likelihood_of_distribution << " ";

            log_likelihood += likelihood_of_distribution;
        }
        // std::cout << std::endl;

        return log_likelihood;
    }

    struct log_likelihood_op : public thrust::unary_function<AssociatedPoint,double> {
        __const__ double log_pi_ij;
        __const__ double base;
        __const__ DevicePoint mu;
        __const__ int index_of_distribution;
        double inv_of_cov[9];

        log_likelihood_op(int index_of_distribution, const WeightedGaussian& distribution) :
            log_pi_ij(distribution.get_weight()),
            base(1.0 / sqrt(pow(2*M_PI, 3) * distribution.get_sigma().det())),
            mu(distribution.get_mu()),
            index_of_distribution(index_of_distribution),
            inv_of_cov {0.0} {
            std::array<double,9> inv_of_sigma = distribution.get_sigma().inverse();

            for(int i=0; i < 9; i++) {
                inv_of_cov[i] = inv_of_sigma[i];
            }
        }

        __host__ __device__
        double operator()(AssociatedPoint p) {
            if(p.likelihoods[index_of_distribution] < 1e-30) {
                return 0.0;
            } else {
                return p.likelihoods[index_of_distribution] * (log_pi_ij + log(likelihood_of_point(p)));
            }
        }

    private:
        __host__ __device__
        double likelihood_of_point(DevicePoint p) {
            DevicePoint x_minus_mu = p - mu;

            double temp_product[3] = {x_minus_mu.z*inv_of_cov[6] + x_minus_mu.y*inv_of_cov[3] + x_minus_mu.x*inv_of_cov[0],
                                      x_minus_mu.z*inv_of_cov[7] + x_minus_mu.y*inv_of_cov[4] + x_minus_mu.x*inv_of_cov[1],
                                      x_minus_mu.z*inv_of_cov[8] + x_minus_mu.y*inv_of_cov[5] + x_minus_mu.x*inv_of_cov[2]};

            double scale_product = x_minus_mu.x * temp_product[0] +
                x_minus_mu.y * temp_product[1] +
                x_minus_mu.z * temp_product[2];

            return base * exp(-0.5 * scale_product);
        }
    };

    double DevicePointCloud::log_likelihood_of_distribution(int index_of_distribution, const WeightedGaussian& distribution) const  {
        return thrust::transform_reduce(pts_begin, pts_end,
                                        log_likelihood_op(index_of_distribution, distribution),
                                        0.0, thrust::plus<double>());
    }

    std::vector<Point> DevicePointCloud::copy_of_points() const {
        std::vector<Point> copy;

        for(auto it = pts_begin; it != pts_end; it++) {
            AssociatedPoint device_point(*it);
            copy.push_back(device_point.to_host());
        }

        return copy;
    }

    HierarchicalGaussianMixture DevicePointCloud::create_hgmm() {
        VLOG(10) << "Creating hgmm...";
        GaussianMixtureFactory gmm_factory;

        std::shared_ptr<DeviceHierarchicalGaussianMixture> hierarchical_mixture(new DeviceHierarchicalGaussianMixture(*this, gmm_factory.from_pcl_corners(*this)));
        hierarchical_mixture->create_children();

        VLOG(10) << "Done creating hgmm.";
        return HierarchicalGaussianMixture(hierarchical_mixture);
    }

    DevicePointCloud::PointIterator DevicePointCloud::begin() {
        return pts_begin;
    }

    DevicePointCloud::PointIterator DevicePointCloud::end() {
        return pts_end;
    }
}
