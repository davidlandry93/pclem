#include "hip/hip_runtime.h"

#include <glog/logging.h>
#include <armadillo>
#include "covariance_matrix.h"

namespace pclem {
    CovarianceMatrix::CovarianceMatrix() : values {{1.0,0.0,0.0,0.0,1.0,0.0,0.0,0.0,1.0}} {}

    CovarianceMatrix::CovarianceMatrix(const CovarianceMatrix& other) {
        values = other.values;
    }

    CovarianceMatrix::CovarianceMatrix(std::array<double,9> _values) {
        for(int i = 0; i < 9; i++) {
            values[i] = _values[i];
        }
    }

    CovarianceMatrix CovarianceMatrix::zeros() {
        std::array<double,9> zeros = {0.0};
        return CovarianceMatrix(zeros);
    }

    void CovarianceMatrix::operator+=(const CovarianceMatrix& rhs) {
        for(int i = 0; i < 9; i++) {
            values[i] += rhs.values[i];
        }
    }

    double CovarianceMatrix::get(int i, int j) const {
        return values[i*3 + j];
    }

    void CovarianceMatrix::set(int i, int j, double new_value) {
        values[i*3 + j] = new_value;
    }

    double CovarianceMatrix::det() const {
        return get(0,0) * (get(1,1) * get(2,2) - get(2, 1) * get(1,2)) -
            get(0,1) * (get(1,0) * get(2,2) - get(1,2) * get(2,0)) +
            get(0,2) * (get(1,0) * get(2,1) + get(2,0) * get(1,1));
    }

    std::array<double,9> CovarianceMatrix::as_array() const {
        auto m = values;
        return m;
    }

    std::array<double,9> CovarianceMatrix::inverse() const {
        VLOG(10) << "Inverting matrix...";

        arma::mat33 arma_cov_mat(values.data());
        arma::mat33 arma_inv_of_cov = arma::inv(arma_cov_mat);

        std::array<double,9> inverse;
        for(auto i = 0; i < 3; i++) {
            for(auto j = 0; j < 3; j++) {
                inverse[i*3 + j] = arma_inv_of_cov(i,j);
            }
        }

        VLOG(10) << "Done inverting matrix.";
        return inverse;
    }

    std::array<double,3> CovarianceMatrix::eigenvalues() const {
        VLOG(10) << "Extracting eigenvalues...";

        arma::mat33 arma_cov_mat(values.data());
        arma::cx_vec eigvals;

        arma::eig_gen(eigvals, arma_cov_mat);

        std::array<double,3> eigvals_array;

        for(int i=0; i < 3; i++) {
            eigvals_array[i] = eigvals[i].real();

            if(std::abs(eigvals[i].imag()) > 1e-300) {
                LOG(WARNING) << "Imaginary part was found during eigenvalue extraction";
            }
        }

        return eigvals_array;
    }

    std::ostream& operator<<(std::ostream& os, const CovarianceMatrix& m) {
        os << "[";
        for(auto value : m.as_array()) {
            os << value << ",";
        }
        os << "]";

        return os;
    }
}
