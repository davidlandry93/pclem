#include "hip/hip_runtime.h"

#include <cstdio>
#include <chrono>
#include <limits>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#include "pointcloud.h"
#include "point.h"

using namespace std::chrono;

namespace pclem {

    PointCloud::PointCloud(std::vector<Point> data, int npoints) :
        data(data), n_points(npoints) {
        updateBoundingBox();
    }

    PointCloud::PointCloud(PointCloud&& other) :
        n_points(other.n_points), boundingBox(other.boundingBox) {
        data = std::move(data);
    }

    PointCloud& PointCloud::operator=(PointCloud&& other) {
        std::swap(data, other.data);
        return *this;
    }

    PointCloud PointCloud::from_vtk(vtkPolyData* vtkData) {
        vtkIdType npoints = vtkData->GetNumberOfPoints();
        vtkPoints* points = vtkData->GetPoints();

        auto stl_vec = std::vector<Point>();
        for(int i=0; i < npoints; i++) {
            double currentPoint[3];

            points->GetPoint(i, currentPoint);
            stl_vec.push_back(Point(currentPoint[0], currentPoint[1], currentPoint[2]));
        }

        return PointCloud(stl_vec, npoints);
    }

    BoundingBox PointCloud::getBoundingBox() {
        return boundingBox;
    }

    struct min_op: public thrust::binary_function<Point,Point,Point> {
        __device__
        Point operator()(Point lhs, Point rhs) {
            return Point(thrust::min(lhs.x, rhs.x),
                         thrust::min(lhs.y, rhs.y),
                         thrust::min(lhs.z, rhs.z));
        }
    };

    struct max_op: public thrust::binary_function<Point,Point,Point> {
        __device__
        Point operator()(Point lhs, Point rhs) {
            return Point(thrust::max(lhs.x, rhs.x),
                         thrust::max(lhs.y, rhs.y),
                         thrust::max(lhs.z, rhs.z));
        }
    };

    void PointCloud::updateBoundingBox(){
        min_op min_function;
        max_op max_function;

        Point min = thrust::reduce(data.begin(), data.end(), Point(0.0,0.0,0.0), min_function);
        Point max = thrust::reduce(data.begin(), data.end(), Point(0.0,0.0,0.0), max_function);

        boundingBox.setMin(min);
        boundingBox.setMax(max);
    }

    void PointCloud::likelihoods(const GaussianMixture& mixture, thrust::device_vector<double>& result) {
        int n_gaussians = mixture.n_gaussians();
        int i = 0;
        for(auto gaussian : mixture) {
            WeightedGaussian host_gaussian = gaussian;
            std::cout << "Sigma in mixture" << host_gaussian.get_sigma().det();

            likelihoods_of_distribution(gaussian, result.begin() + i*n_gaussians);
            i++;
        }
    }

    class gaussian_op : public thrust::unary_function<Point,double> {
    public:
        __host__
        gaussian_op(const WeightedGaussian& _g) : g(_g) {
            det_sigma = g.get_sigma().det();
        }
        __device__
        double operator()(Point p) {
            //printf("%d", det_sigma);
            return det_sigma;
        }

    private:
        WeightedGaussian g;
        double det_sigma;
    };

    void PointCloud::likelihoods_of_distribution(WeightedGaussian gaussian,
                                                 thrust::device_vector<double>::iterator result) {
        gaussian_op op(gaussian);

        //thrust::transform(data.begin(), data.end(), result, op);
    }

    int PointCloud::get_n_points() const {
        return n_points;
    }
}
