#include "hip/hip_runtime.h"

#include <math.h>
#include <cstdio>
#include <chrono>
#include <limits>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#include <armadillo>

#include "strided_range.h"
#include "pointcloud.h"
#include "point.cuh"

using namespace std::chrono;

namespace pclem {

    PointCloud::PointCloud(std::vector<Point> data, int npoints) :
        data(data), n_points(npoints) {
        updateBoundingBox();
    }

    PointCloud::PointCloud(PointCloud&& other) :
        data(), n_points(other.n_points), boundingBox(other.boundingBox) {
        std::swap(data,other.data);
    }

    PointCloud& PointCloud::operator=(PointCloud&& other) {
        std::swap(data, other.data);
        return *this;
    }

    PointCloud PointCloud::from_vtk(vtkPolyData* vtkData) {
        vtkIdType npoints = vtkData->GetNumberOfPoints();
        vtkPoints* points = vtkData->GetPoints();

        auto stl_vec = std::vector<Point>();
        for(int i=0; i < npoints; i++) {
            double currentPoint[3];

            points->GetPoint(i, currentPoint);
            stl_vec.push_back(Point(currentPoint[0], currentPoint[1], currentPoint[2]));
        }

        return PointCloud(stl_vec, npoints);
    }

    BoundingBox PointCloud::getBoundingBox() {
        return boundingBox;
    }

    struct min_op: public thrust::binary_function<Point,Point,Point> {
        __device__
        Point operator()(Point lhs, Point rhs) {
            return Point(thrust::min(lhs.x, rhs.x),
                         thrust::min(lhs.y, rhs.y),
                         thrust::min(lhs.z, rhs.z));
        }
    };

    struct max_op: public thrust::binary_function<Point,Point,Point> {
        __device__
        Point operator()(Point lhs, Point rhs) {
            return Point(thrust::max(lhs.x, rhs.x),
                         thrust::max(lhs.y, rhs.y),
                         thrust::max(lhs.z, rhs.z));
        }
    };

    void PointCloud::updateBoundingBox(){
        min_op min_function;
        max_op max_function;

        Point min = thrust::reduce(data.begin(), data.end(), Point(0.0,0.0,0.0), min_function);
        Point max = thrust::reduce(data.begin(), data.end(), Point(0.0,0.0,0.0), max_function);

        boundingBox.setMin(min);
        boundingBox.setMax(max);
    }

    void PointCloud::likelihoods(const GaussianMixture& mixture, thrust::device_vector<double>& result) const {
        int n_gaussians = mixture.n_gaussians();

        for(int i=0; i < n_gaussians; i++) {
            WeightedGaussian g = mixture.get_gaussian(i);

            likelihoods_of_distribution(mixture.get_gaussian(i), result.begin() + i*n_points);
            normalize_likelihoods(result, n_gaussians, n_points);
        }
    }

    struct gaussian_op : public thrust::unary_function<Point,double> {
    public:
        gaussian_op(Point _mu, double _det, std::array<double,9> _inv_of_covariance) :
            mu(_mu),
            base(1 / sqrt(pow(2*M_PI, 3) * _det)),
            inv() {
            for(int i = 0; i < 9; i++) {
                inv[i] = _inv_of_covariance[i];
            }


            std::cout << _mu;
            std::cout << _det;
        }


        // https://en.wikipedia.org/wiki/Multivariate_normal_distribution#Density_function
        __host__ __device__
        double operator()(Point x) {
            Point x_minus_mu = x - mu;
            double temp_product[3] = {x_minus_mu.z*inv[6] + x_minus_mu.y*inv[3] + x_minus_mu.x*inv[0],
                                      x_minus_mu.z*inv[7] + x_minus_mu.y*inv[4] + x_minus_mu.x*inv[1],
                                      x_minus_mu.z*inv[8] + x_minus_mu.y*inv[5] + x_minus_mu.x*inv[2]};

            double scale_product =
                x_minus_mu.x*temp_product[0] +
                x_minus_mu.y*temp_product[1] +
                x_minus_mu.z*temp_product[2];

            return base * exp(-0.5 * scale_product);
        }

        __const__ Point mu;
        __const__ double base;
        double inv[9];
    };

    void PointCloud::likelihoods_of_distribution(WeightedGaussian gaussian,
                                                 thrust::device_vector<double>::iterator result) const {
        std::array<double,9> cov_mat = gaussian.get_sigma().as_array();

        arma::mat33 arma_cov_mat(cov_mat.data());
        double det_of_covariance = arma::det(arma_cov_mat);

        arma::mat33 arma_inv_of_covariance = arma::inv_sympd(arma_cov_mat);

        std::array<double,9> inv_of_covariance;
        for(auto i = 0; i < 3; i++) {
            for (auto j=0; j < 3; j++) {
                inv_of_covariance[i*3+j] = arma_inv_of_covariance(i,j);
            }
        }

        gaussian_op op(gaussian.get_mu(), det_of_covariance, inv_of_covariance);

        std::cout << "callin" << std::endl;
        thrust::transform(data.begin(), data.end(), result, op);

        std::cout.precision(17);
        std::cout << std::scientific;
        std::cout << *result << " " << *(result + 1) << std::endl;
        std::cout << "Max: " << *(thrust::max_element(result, result + (data.end() - data.begin()))) << std::endl;
    }

    struct normalization_op : public thrust::unary_function<double,double> {
        double operator()(double likelihood) {
            return 0.0;
        }
    };

    void PointCloud::normalize_likelihoods(thrust::device_vector<double>& likelihoods,
                                           int n_gaussians,
                                           int n_points) const {
        typedef thrust::device_vector<double>::iterator Iterator;

        for(int i = 0; i < n_points; i++) {
            StridedRange<Iterator> range(likelihoods.begin() + i, likelihoods.end(), n_points);

            double sum = thrust::reduce(range.begin(), range.end(), 0.0, thrust::plus<double>());
            std::cout << sum << std::endl;
        }
    }

    int PointCloud::get_n_points() const {
        return n_points;
    }
}
