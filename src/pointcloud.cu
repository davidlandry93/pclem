#include "hip/hip_runtime.h"

#include <math.h>
#include <cstdio>
#include <chrono>
#include <limits>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#include <armadillo>

#include "strided_range.h"
#include "pointcloud.h"
#include "point.cuh"

using namespace std::chrono;

namespace pclem {

    PointCloud::PointCloud(std::vector<Point> data, int npoints) :
        data(data), n_points(npoints) {
        updateBoundingBox();
    }

    PointCloud::PointCloud(PointCloud&& other) :
        data(), n_points(other.n_points), boundingBox(other.boundingBox) {
        std::swap(data,other.data);
    }

    PointCloud& PointCloud::operator=(PointCloud&& other) {
        std::swap(data, other.data);
        return *this;
    }

    PointCloud PointCloud::from_vtk(vtkPolyData* vtkData) {
        vtkIdType npoints = vtkData->GetNumberOfPoints();
        vtkPoints* points = vtkData->GetPoints();

        auto stl_vec = std::vector<Point>();
        for(int i=0; i < npoints; i++) {
            double currentPoint[3];

            points->GetPoint(i, currentPoint);
            stl_vec.push_back(Point(currentPoint[0], currentPoint[1], currentPoint[2]));
        }

        return PointCloud(stl_vec, npoints);
    }

    BoundingBox PointCloud::getBoundingBox() {
        return boundingBox;
    }

    struct min_op: public thrust::binary_function<Point,Point,Point> {
        __device__
        Point operator()(Point lhs, Point rhs) {
            return Point(thrust::min(lhs.x, rhs.x),
                         thrust::min(lhs.y, rhs.y),
                         thrust::min(lhs.z, rhs.z));
        }
    };

    struct max_op: public thrust::binary_function<Point,Point,Point> {
        __device__
        Point operator()(Point lhs, Point rhs) {
            return Point(thrust::max(lhs.x, rhs.x),
                         thrust::max(lhs.y, rhs.y),
                         thrust::max(lhs.z, rhs.z));
        }
    };

    void PointCloud::updateBoundingBox(){
        min_op min_function;
        max_op max_function;

        Point min = thrust::reduce(data.begin(), data.end(), Point(0.0,0.0,0.0), min_function);
        Point max = thrust::reduce(data.begin(), data.end(), Point(0.0,0.0,0.0), max_function);

        boundingBox.setMin(min);
        boundingBox.setMax(max);
    }


    struct normalization_op : public thrust::unary_function<double,double> {
        double operator()(double likelihood) {
            return 0.0;
        }
    };

    thrust::device_vector<Point>::const_iterator PointCloud::begin() const {
        return data.begin();
    }

    thrust::device_vector<Point>::const_iterator PointCloud::end() const {
        return data.end();
    }

    void PointCloud::normalize_likelihoods(thrust::device_vector<double>& likelihoods,
                                           int n_gaussians,
                                           int n_points) const {
        typedef thrust::device_vector<double>::iterator Iterator;

        for(int i = 0; i < n_points; i++) {
            StridedRange<Iterator> range(likelihoods.begin() + i, likelihoods.end(), n_points);

            double sum = thrust::reduce(range.begin(), range.end(), 0.0, thrust::plus<double>());
            std::cout << sum << std::endl;
        }
    }

    int PointCloud::get_n_points() const {
        return n_points;
    }
}
