#include "hip/hip_runtime.h"

#include <chrono>
#include <limits>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#include "pointcloud.h"

using namespace std::chrono;

namespace pclem {

    PointCloud::PointCloud(std::vector<double> data, int npoints) :
        data(data), n_of_points(npoints) {
        updateBoundingBox();
    }

    PointCloud::PointCloud(PointCloud&& other) :
        n_of_points(other.n_of_points), boundingBox(other.boundingBox) {
        data = std::move(data);
    }

    PointCloud& PointCloud::operator=(PointCloud&& other) {
        std::swap(data, other.data);
        return *this;
    }

    PointCloud PointCloud::from_vtk(vtkPolyData* vtkData) {
        vtkIdType npoints = vtkData->GetNumberOfPoints();
        vtkPoints* points = vtkData->GetPoints();

        auto stl_vec = std::vector<double>(3*npoints);
        for(int i=0; i < npoints; i++) {
            double currentPoint[3];

            points->GetPoint(i, currentPoint);
            stl_vec[i] = currentPoint[0];
            stl_vec[i + npoints] = currentPoint[1];
            stl_vec[i + 2*npoints] = currentPoint[2];
        }

        return PointCloud(stl_vec, npoints);
    }

    BoundingBox PointCloud::getBoundingBox() {
        return boundingBox;
    }

    void PointCloud::updateBoundingBox(){
        double min[3];
        double max[3];

        for(int i=0; i < 3; i++) {
            auto found_max = thrust::max_element(data.begin() + i*n_of_points,
                                                 data.begin() + (i + 1)*n_of_points);
            auto found_min = thrust::min_element(data.begin() + i*n_of_points,
                                                 data.begin() + (i + 1)*n_of_points);
            max[i] = *found_max;
            min[i] = *found_min;
        }

        boundingBox.setMin(Point(min[0], min[1], min[2]));
        boundingBox.setMax(Point(max[0], max[1], max[2]));
    }
}
