#include "hip/hip_runtime.h"

#include <math.h>
#include <cstdio>
#include <chrono>
#include <limits>
#include <glog/logging.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#include <armadillo>

#include "strided_range.h"
#include "point.cuh"
#include "pointcloud.h"
#include "raw_covariance_matrix.h"

using namespace std::chrono;

namespace pclem {

    PointCloud::PointCloud(std::vector<AssociatedPoint> data) :
        data(data), n_points(data.size()) {
        updateBoundingBox();
    }

    PointCloud::PointCloud(PointCloud&& other) :
        data(), n_points(other.n_points), boundingBox(other.boundingBox) {
        std::swap(data,other.data);
    }

    PointCloud& PointCloud::operator=(PointCloud&& other) {
        std::swap(data, other.data);
        return *this;
    }

    PointCloud PointCloud::from_vtk(vtkPolyData* vtkData) {
        VLOG(2) << "Building point cloud...";

        vtkIdType npoints = vtkData->GetNumberOfPoints();
        vtkPoints* points = vtkData->GetPoints();

        auto stl_vec = std::vector<AssociatedPoint>();
        for(int i=0; i < npoints; i++) {
            double currentPoint[3];

            points->GetPoint(i, currentPoint);
            stl_vec.push_back(AssociatedPoint(currentPoint[0], currentPoint[1], currentPoint[2]));
        }

        VLOG(2) << "Done.";
        return PointCloud(stl_vec);
    }

    BoundingBox PointCloud::getBoundingBox() {
        return boundingBox;
    }

    struct min_op: public thrust::binary_function<Point,Point,Point> {
        __device__
        Point operator()(Point lhs, Point rhs) {
            return Point(thrust::min(lhs.x, rhs.x),
                         thrust::min(lhs.y, rhs.y),
                         thrust::min(lhs.z, rhs.z));
        }
    };

    struct max_op: public thrust::binary_function<Point,Point,Point> {
        __device__
        Point operator()(Point lhs, Point rhs) {
            return Point(thrust::max(lhs.x, rhs.x),
                         thrust::max(lhs.y, rhs.y),
                         thrust::max(lhs.z, rhs.z));
        }
    };

    void PointCloud::updateBoundingBox(){
        min_op min_function;
        max_op max_function;

        Point min = thrust::reduce(data.begin(), data.end(), Point(0.0,0.0,0.0), min_function);
        Point max = thrust::reduce(data.begin(), data.end(), Point(0.0,0.0,0.0), max_function);

        boundingBox.setMin(min);
        boundingBox.setMax(max);
    }

    thrust::device_vector<AssociatedPoint>::const_iterator PointCloud::begin() const {
        return data.begin();
    }

    thrust::device_vector<AssociatedPoint>::const_iterator PointCloud::end() const {
        return data.end();
    }

    int PointCloud::get_n_points() const {
        return n_points;
    }

    void PointCloud::compute_associations(const GaussianMixture& mixture) {
        VLOG(10) << "Computing point/distribution associations...";

        for(int i = 0; i < mixture.n_gaussians(); i++) {
            compute_associations_of_distribution(i, mixture.get_gaussian(i));
        }

        VLOG(10) << "Done computing point/distribution associations.";
    }

    struct gaussian_op : public thrust::unary_function<AssociatedPoint,AssociatedPoint> {
    public:
        gaussian_op(int index_of_distribution, const WeightedGaussian& distribution) :
            index_of_distribution(index_of_distribution),
            mu(distribution.get_mu()),
            base(distribution.get_weight() / sqrt(pow(2*M_PI,3) * distribution.get_sigma().det())),
            inv_of_covariance {0.0} {
            std::array<double,9> computed_inv = distribution.get_sigma().inverse();

            for (int i = 0; i < 9; i++) {
                inv_of_covariance[i] = computed_inv[i];
            }
        }

        __host__ __device__
        AssociatedPoint operator()(AssociatedPoint p) {
            p.likelihoods[index_of_distribution] = likelihood_of_point(p);
            return p;
        }

    private:
        __const__ int index_of_distribution;
        __const__ Point mu;
        __const__ double base;
        double inv_of_covariance[9];

        __host__ __device__
        double likelihood_of_point(Point p) {
            Point x_minus_mu = p - mu;

            double temp_product[3] = {x_minus_mu.z*inv_of_covariance[6] + x_minus_mu.y*inv_of_covariance[3] + x_minus_mu.x*inv_of_covariance[0],
                                      x_minus_mu.z*inv_of_covariance[7] + x_minus_mu.y*inv_of_covariance[4] + x_minus_mu.x*inv_of_covariance[1],
                                      x_minus_mu.z*inv_of_covariance[8] + x_minus_mu.y*inv_of_covariance[5] + x_minus_mu.x*inv_of_covariance[2]};

            double scale_product = x_minus_mu.x * temp_product[0] +
                x_minus_mu.y * temp_product[1] +
                x_minus_mu.z * temp_product[2];

            return base * exp(-0.5 * scale_product);
        }
    };

    void PointCloud::compute_associations_of_distribution(int index_of_distribution, const WeightedGaussian& distribution) {
        gaussian_op op(index_of_distribution, distribution);

        thrust::transform(data.begin(), data.end(), data.begin(), op);
    }

    struct normalization_op : public thrust::unary_function<AssociatedPoint, AssociatedPoint> {
        __host__ __device__
        AssociatedPoint operator()(AssociatedPoint p) {
            double sum_of_associations = 0.0;
            for(int i = 0; i < p.N_DISTRIBUTIONS_PER_MIXTURE; i++) {
                sum_of_associations += p.likelihoods[i];
            }

            for(int i = 0; i < p.N_DISTRIBUTIONS_PER_MIXTURE; i++) {
                p.likelihoods[i] = p.likelihoods[i] / sum_of_associations;
            }

            return p;
        }
    };

    void PointCloud::normalize_associations() {
        VLOG(10) << "Normalizing associations for every point...";

        thrust::transform(data.begin(), data.end(), data.begin(), normalization_op());

        VLOG(10) << "Done normalizing associations";
    }

    // This functor stores the sums of gammas in an AssociatedPoint.
    struct sums_of_gammas_op : public thrust::binary_function<AssociatedPoint, AssociatedPoint, AssociatedPoint> {

        __host__ __device__
        AssociatedPoint operator()(AssociatedPoint lhs, AssociatedPoint rhs) {
            AssociatedPoint sum;
            for(int i=0; i < 8; i++) {
                sum.likelihoods[i] = lhs.likelihoods[i] + rhs.likelihoods[i];
            }
            return sum;
        }
    };

    GaussianMixture PointCloud::create_mixture() const {
        // We store the sum of gammas of every distribution in an empty, meaningless AssociatedPoint.
        AssociatedPoint sums;
        sums = thrust::reduce(data.begin(), data.end(), AssociatedPoint(), sums_of_gammas_op());

        std::vector<WeightedGaussian> gaussians;
        for(int i=0; i < AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE; i++) {
            gaussians.push_back(create_distribution_of_mixture(i, sums.likelihoods[i]));
        }

        return GaussianMixture(gaussians);
    }

    struct weight_point_op : public thrust::unary_function<AssociatedPoint,Point> {
        int index_of_distribution;

        weight_point_op(int index_of_distribution) : index_of_distribution(index_of_distribution) {}

        __host__ __device__
        Point operator()(AssociatedPoint p) {
            return Point(p.x * p.likelihoods[index_of_distribution],
                         p.y * p.likelihoods[index_of_distribution],
                         p.z * p.likelihoods[index_of_distribution]);
        }
    };

    struct sum_of_points_op : public thrust::binary_function<Point,Point,Point> {
        __host__ __device__
        Point operator()(Point lhs, Point rhs) {
            return lhs + rhs;
        }
    };

    WeightedGaussian PointCloud::create_distribution_of_mixture(int index_of_distribution, double sum_of_gammas) const {
        VLOG(10) << "Creating distribution " << index_of_distribution << " of mixture...";

        Point new_mu = thrust::transform_reduce(data.begin(), data.end(),
                                                weight_point_op(index_of_distribution),
                                                Point(0.0, 0.0, 0.0),
                                                sum_of_points_op());

        new_mu = Point(new_mu.x / sum_of_gammas,
                       new_mu.y / sum_of_gammas,
                       new_mu.z / sum_of_gammas);

        VLOG(1) << "New mu: " << new_mu;

        CovarianceMatrix new_sigma = compute_sigma(index_of_distribution, new_mu, sum_of_gammas);

        double new_weight = sum_of_gammas / get_n_points();

        VLOG(10) << "Done creating distribution " << index_of_distribution << " of mixture.";
        return WeightedGaussian(new_mu, new_sigma, new_weight);
    }

    struct point_to_cov_op : public thrust::unary_function<AssociatedPoint, RawCovarianceMatrix> {
        int index_of_distribution;

        point_to_cov_op(int index_of_distribution) : index_of_distribution(index_of_distribution) {}

        __host__ __device__
        RawCovarianceMatrix operator()(AssociatedPoint p) {
            RawCovarianceMatrix r;
            double gamma = p.likelihoods[index_of_distribution];

            r.v00 = p.x * p.x * gamma;
            r.v11 = p.y * p.y * gamma;
            r.v22 = p.z * p.z * gamma;

            r.v10 = r.v01 = p.x * p.y * gamma;
            r.v20 = r.v02 = p.x * p.z * gamma;
            r.v21 = r.v12 = p.y * p.z * gamma;

            return r;
        }
    };

    struct sum_of_cov_op : public thrust::binary_function<RawCovarianceMatrix, RawCovarianceMatrix, RawCovarianceMatrix> {
        __host__ __device__
        RawCovarianceMatrix operator()(RawCovarianceMatrix lhs, RawCovarianceMatrix rhs) {
            return lhs + rhs;
        }
    };

    CovarianceMatrix PointCloud::compute_sigma(int index_of_distribution, const Point& mu, double sum_of_gammas) const {
        RawCovarianceMatrix init = RawCovarianceMatrix::zeros();

        RawCovarianceMatrix sigma = thrust::transform_reduce(data.begin(), data.end(),
                                                             point_to_cov_op(index_of_distribution),
                                                             init, sum_of_cov_op());

        RawCovarianceMatrix base_sigma;

        base_sigma.v00 = mu.x * mu.x;
        base_sigma.v11 = mu.y * mu.y;
        base_sigma.v22 = mu.z * mu.z;

        base_sigma.v01 = base_sigma.v10 = mu.x * mu.y;
        base_sigma.v02 = base_sigma.v20 = mu.x * mu.z;
        base_sigma.v21 = base_sigma.v12 = mu.z * mu.y;

        sigma = sigma / sum_of_gammas - base_sigma;
        return CovarianceMatrix(sigma);
    }

    double PointCloud::log_likelihood_of_mixture(const GaussianMixture& mixture) const {
        double log_likelihood = 0.0;

        std::cout << "Log likelihood of distributions: ";
        for(int i=0; i < AssociatedPoint::N_DISTRIBUTIONS_PER_MIXTURE; i++) {
            double likelihood_of_distribution = log_likelihood_of_distribution(i, mixture.get_gaussian(i));

            std::cout << likelihood_of_distribution << " ";

            log_likelihood += likelihood_of_distribution;
        }
        std::cout << std::endl;

        return log_likelihood;
    }

    struct log_likelihood_op : public thrust::unary_function<AssociatedPoint,double> {
        __const__ double log_pi_ij;
        __const__ double base;
        __const__ Point mu;
        __const__ int index_of_distribution;
        double inv_of_cov[9];

        log_likelihood_op(int index_of_distribution, const WeightedGaussian& distribution) :
            log_pi_ij(distribution.get_weight()),
            base(1.0 / sqrt(pow(2*M_PI, 3) * distribution.get_sigma().det())),
            mu(distribution.get_mu()),
            index_of_distribution(index_of_distribution),
            inv_of_cov {0.0} {
            std::array<double,9> inv_of_sigma = distribution.get_sigma().inverse();

            for(int i=0; i < 9; i++) {
                inv_of_cov[i] = inv_of_sigma[i];
            }
        }

        __host__ __device__
        double operator()(AssociatedPoint p) {
            if(p.likelihoods[index_of_distribution] < 1e-70) {
                return 0.0;
            } else {
                return p.likelihoods[index_of_distribution] * (log_pi_ij + log(likelihood_of_point(p)));
            }
        }

    private:
        __host__ __device__
        double likelihood_of_point(Point p) {
            Point x_minus_mu = p - mu;

            double temp_product[3] = {x_minus_mu.z*inv_of_cov[6] + x_minus_mu.y*inv_of_cov[3] + x_minus_mu.x*inv_of_cov[0],
                                      x_minus_mu.z*inv_of_cov[7] + x_minus_mu.y*inv_of_cov[4] + x_minus_mu.x*inv_of_cov[1],
                                      x_minus_mu.z*inv_of_cov[8] + x_minus_mu.y*inv_of_cov[5] + x_minus_mu.x*inv_of_cov[2]};

            double scale_product = x_minus_mu.x * temp_product[0] +
                x_minus_mu.y * temp_product[1] +
                x_minus_mu.z * temp_product[2];

            return base * exp(-0.5 * scale_product);
        }
    };

    double PointCloud::log_likelihood_of_distribution(int index_of_distribution, const WeightedGaussian& distribution) const  {
        return thrust::transform_reduce(data.begin(), data.end(),
                                        log_likelihood_op(index_of_distribution, distribution),
                                        0.0, thrust::plus<double>());
    }
}
